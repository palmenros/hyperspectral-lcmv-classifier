//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: native_multi_target_detection_ldl_initialize.cu
//
// GPU Coder version                    : 24.1
// CUDA/C/C++ source code generated on  : 17-May-2024 07:49:02
//

// Include Files
#include "native_multi_target_detection_ldl_initialize.h"
#include "native_multi_target_detection_ldl_data.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"
#include "MWMemoryManager.hpp"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void native_multi_target_detection_ldl_initialize()
{
  hipGetLastError();
  mwMemoryManagerInit(256U, 0U, 8U, 2048U);
  cublasEnsureInitialization(HIPBLAS_POINTER_MODE_HOST);
  cusolverEnsureInitialization();
  isInitialized_native_multi_target_detection_ldl = true;
}

//
// File trailer for native_multi_target_detection_ldl_initialize.cu
//
// [EOF]
//

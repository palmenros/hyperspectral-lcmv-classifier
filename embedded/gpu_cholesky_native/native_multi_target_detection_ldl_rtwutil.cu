//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: native_multi_target_detection_ldl_rtwutil.cu
//
// GPU Coder version                    : 24.1
// CUDA/C/C++ source code generated on  : 17-May-2024 07:49:02
//

// Include Files
#include "native_multi_target_detection_ldl_rtwutil.h"
#include "rt_nonfinite.h"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"

// Function Definitions
//
// Arguments    : int errCode
//                const char *file
//                unsigned int b_line
//                const char *errorName
//                const char *errorString
// Return Type  : void
//
void b_raiseCudaError(int errCode, const char *file, unsigned int b_line,
                      const char *errorName, const char *errorString)
{
  printf("ERR[%d] %s:%s in file %s at line %d\nExiting program execution ...\n",
         errCode, errorName, errorString, file, b_line);
  exit(errCode);
}

//
// Arguments    : hipError_t errCode
//                const char *file
//                unsigned int b_line
// Return Type  : void
//
void checkCudaError(hipError_t errCode, const char *file, unsigned int b_line)
{
  if (errCode != hipSuccess) {
    b_raiseCudaError(errCode, file, b_line, hipGetErrorName(errCode),
                     hipGetErrorString(errCode));
  }
}

//
// File trailer for native_multi_target_detection_ldl_rtwutil.cu
//
// [EOF]
//

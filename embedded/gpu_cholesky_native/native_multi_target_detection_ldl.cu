#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: native_multi_target_detection_ldl.cu
//
// GPU Coder version                    : 24.1
// CUDA/C/C++ source code generated on  : 17-May-2024 07:49:02
//

// Include Files
#include "native_multi_target_detection_ldl.h"
#include "native_multi_target_detection_ldl_data.h"
#include "native_multi_target_detection_ldl_initialize.h"
#include "native_multi_target_detection_ldl_rtwutil.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWErrorCodeUtils.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "hip/hip_math_constants.h"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include <time.h>

// Function Declarations
static
#ifdef __HIPCC__
    __device__
#endif
    long
    computeEndIdx_device(long start, long end, long stride);

static unsigned long computeNumIters(int ub);

static void cublasCheck(hipblasStatus_t errCode, const char *file,
                        unsigned int b_line);

static void cusolverCheck(hipsolverStatus_t errCode, const char *file,
                          unsigned int b_line);

static __global__ void
native_multi_target_detection_ldl_kernel1(const float X[692224],
                                          float a[692224]);

static __global__ void
native_multi_target_detection_ldl_kernel10(float t2[225]);

static __global__ void
native_multi_target_detection_ldl_kernel11(const int info_t, const int b,
                                           float t2[225]);

static __global__ void
native_multi_target_detection_ldl_kernel12(const float t2[225], float A[225]);

static __global__ void
native_multi_target_detection_ldl_kernel13(const float C[45], float W[45]);

static __global__ void native_multi_target_detection_ldl_kernel14(float W[45]);

static __global__ void native_multi_target_detection_ldl_kernel15(float W[45]);

static __global__ void
native_multi_target_detection_ldl_kernel2(float R[28561]);

static __global__ void
native_multi_target_detection_ldl_kernel3(float R[28561]);

static __global__ void
native_multi_target_detection_ldl_kernel4(const int info_t, const int b,
                                          float R[28561]);

static __global__ void
native_multi_target_detection_ldl_kernel5(const float R[28561], float A[28561]);

static __global__ void
native_multi_target_detection_ldl_kernel6(const float T[2535], float W[2535]);

static __global__ void native_multi_target_detection_ldl_kernel7(float W[2535]);

static __global__ void native_multi_target_detection_ldl_kernel8(float W[2535]);

static __global__ void
native_multi_target_detection_ldl_kernel9(const float T[2535], float a[2535]);

static void raiseCudaError(int errCode, const char *file, unsigned int b_line,
                           const char *errorName, const char *errorString);

// Function Definitions
//
// Arguments    : long start
//                long end
//                long stride
// Return Type  : long
//
static __device__ long computeEndIdx_device(long start, long end, long stride)
{
  long newEnd;
  newEnd = -1L;
  if ((stride > 0L) && (start <= end)) {
    newEnd = (end - start) / stride;
  } else if ((stride < 0L) && (end <= start)) {
    newEnd = (start - end) / -stride;
  }
  return newEnd;
}

//
// Arguments    : int ub
// Return Type  : unsigned long
//
static unsigned long computeNumIters(int ub)
{
  unsigned long numIters;
  numIters = 0UL;
  if (ub >= 0) {
    numIters = static_cast<unsigned long>(ub + 1);
  }
  return numIters;
}

//
// Arguments    : hipblasStatus_t errCode
//                const char *file
//                unsigned int b_line
// Return Type  : void
//
static void cublasCheck(hipblasStatus_t errCode, const char *file,
                        unsigned int b_line)
{
  const char *errName;
  const char *errString;
  if (errCode != HIPBLAS_STATUS_SUCCESS) {
    cublasGetErrorName(errCode, &errName);
    cublasGetErrorString(errCode, &errString);
    raiseCudaError(errCode, file, b_line, errName, errString);
  }
}

//
// Arguments    : hipsolverStatus_t errCode
//                const char *file
//                unsigned int b_line
// Return Type  : void
//
static void cusolverCheck(hipsolverStatus_t errCode, const char *file,
                          unsigned int b_line)
{
  const char *errName;
  const char *errString;
  if (errCode != HIPSOLVER_STATUS_SUCCESS) {
    cusolverGetErrorName(errCode, &errName);
    cusolverGetErrorString(errCode, &errString);
    raiseCudaError(errCode, file, b_line, errName, errString);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float X[692224]
//                float a[692224]
// Return Type  : void
//
static __global__
    __launch_bounds__(512, 1) void native_multi_target_detection_ldl_kernel1(
        const float X[692224], float a[692224])
{
  unsigned long gThreadId;
  int i;
  int j;
  gThreadId = mwGetGlobalThreadIndex();
  j = static_cast<int>(gThreadId % 169UL);
  i = static_cast<int>((gThreadId - static_cast<unsigned long>(j)) / 169UL);
  if ((i < 4096) && (j < 169)) {
    //  R = correlation_matrix(X);
    //  TIMING_0
    a[j + 169 * i] = X[i + (j << 12)];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float t2[225]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void native_multi_target_detection_ldl_kernel10(float t2[225])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 225) {
    t2[i] = HIP_NAN_F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int info_t
//                const int b
//                float t2[225]
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void native_multi_target_detection_ldl_kernel11(
        const int info_t, const int b, float t2[225])
{
  unsigned long gStride;
  unsigned long gThreadId;
  unsigned long loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(b);
  for (unsigned long idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int i;
    int j;
    j = static_cast<int>(idx);
    i = j + 2;
    for (long b_i{0L};
         b_i <= computeEndIdx_device(static_cast<long>(i),
                                     static_cast<long>(info_t), 1L);
         b_i++) {
      t2[(static_cast<int>(static_cast<long>(j + 2) + b_i) + 15 * j) - 1] =
          0.0F;
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float t2[225]
//                float A[225]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void native_multi_target_detection_ldl_kernel12(const float t2[225],
                                                            float A[225])
{
  unsigned long gThreadId;
  int i;
  int j;
  gThreadId = mwGetGlobalThreadIndex();
  j = static_cast<int>(gThreadId % 15UL);
  i = static_cast<int>((gThreadId - static_cast<unsigned long>(j)) / 15UL);
  if ((i < 15) && (j < 15)) {
    A[j + 15 * i] = t2[i + 15 * j];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float C[45]
//                float W[45]
// Return Type  : void
//
static __global__ __launch_bounds__(
    64, 1) void native_multi_target_detection_ldl_kernel13(const float C[45],
                                                           float W[45])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 45) {
    W[i] = C[i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float W[45]
// Return Type  : void
//
static __global__ __launch_bounds__(
    64, 1) void native_multi_target_detection_ldl_kernel14(float W[45])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 45) {
    W[i] = HIP_NAN_F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float W[45]
// Return Type  : void
//
static __global__ __launch_bounds__(
    64, 1) void native_multi_target_detection_ldl_kernel15(float W[45])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 45) {
    W[i] = HIP_NAN_F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float R[28561]
// Return Type  : void
//
static __global__ __launch_bounds__(
    288, 1) void native_multi_target_detection_ldl_kernel2(float R[28561])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 28561) {
    //  TIMING_1
    R[i] /= 4096.0F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float R[28561]
// Return Type  : void
//
static __global__ __launch_bounds__(
    288, 1) void native_multi_target_detection_ldl_kernel3(float R[28561])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 28561) {
    R[i] = HIP_NAN_F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int info_t
//                const int b
//                float R[28561]
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void native_multi_target_detection_ldl_kernel4(
        const int info_t, const int b, float R[28561])
{
  unsigned long gStride;
  unsigned long gThreadId;
  unsigned long loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(b);
  for (unsigned long idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int i;
    int j;
    j = static_cast<int>(idx);
    i = j + 2;
    for (long b_i{0L};
         b_i <= computeEndIdx_device(static_cast<long>(i),
                                     static_cast<long>(info_t), 1L);
         b_i++) {
      R[(static_cast<int>(static_cast<long>(j + 2) + b_i) + 169 * j) - 1] =
          0.0F;
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float R[28561]
//                float A[28561]
// Return Type  : void
//
static __global__ __launch_bounds__(
    288, 1) void native_multi_target_detection_ldl_kernel5(const float R[28561],
                                                           float A[28561])
{
  unsigned long gThreadId;
  int i;
  int j;
  gThreadId = mwGetGlobalThreadIndex();
  j = static_cast<int>(gThreadId % 169UL);
  i = static_cast<int>((gThreadId - static_cast<unsigned long>(j)) / 169UL);
  if ((i < 169) && (j < 169)) {
    A[j + 169 * i] = R[i + 169 * j];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float T[2535]
//                float W[2535]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void native_multi_target_detection_ldl_kernel6(const float T[2535],
                                                           float W[2535])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 2535) {
    W[i] = T[i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float W[2535]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void native_multi_target_detection_ldl_kernel7(float W[2535])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 2535) {
    W[i] = HIP_NAN_F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float W[2535]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void native_multi_target_detection_ldl_kernel8(float W[2535])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 2535) {
    W[i] = HIP_NAN_F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float T[2535]
//                float a[2535]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void native_multi_target_detection_ldl_kernel9(const float T[2535],
                                                           float a[2535])
{
  unsigned long gThreadId;
  int i;
  int j;
  gThreadId = mwGetGlobalThreadIndex();
  j = static_cast<int>(gThreadId % 15UL);
  i = static_cast<int>((gThreadId - static_cast<unsigned long>(j)) / 15UL);
  if ((i < 169) && (j < 15)) {
    //  TIMING_3
    a[j + 15 * i] = T[i + 169 * j];
  }
}

//
// Arguments    : int errCode
//                const char *file
//                unsigned int b_line
//                const char *errorName
//                const char *errorString
// Return Type  : void
//
static void raiseCudaError(int errCode, const char *file, unsigned int b_line,
                           const char *errorName, const char *errorString)
{
  printf("ERR[%d] %s:%s in file %s at line %d\nExiting program execution ...\n",
         errCode, errorName, errorString, file, b_line);
  exit(errCode);
}

//
// Arguments    : const float cpu_T[2535]
//                const float cpu_C[45]
//                const float cpu_X[692224]
//                float cpu_W[507]
// Return Type  : void
//
void native_multi_target_detection_ldl(const float cpu_T[2535],
                                       const float cpu_C[45],
                                       const float cpu_X[692224],
                                       float cpu_W[507])
{
  dim3 block;
  dim3 grid;
  float(*gpu_X)[692224];
  float(*gpu_a)[692224];
  float(*gpu_A)[28561];
  float(*gpu_R)[28561];
  float(*b_gpu_a)[2535];
  float(*gpu_T)[2535];
  float(*gpu_W)[2535];
  float(*c_gpu_W)[507];
  float(*b_gpu_A)[225];
  float(*gpu_t2)[225];
  float(*b_gpu_W)[45];
  float(*gpu_C)[45];
  float alpha1;
  float beta1;
  int(*gpu_IPIV)[169];
  int(*b_gpu_IPIV)[15];
  int b_cpu_info;
  int c_cpu_info;
  int cpu_info;
  int cpu_info_t;
  int d_cpu_info;
  int *b_gpu_info;
  int *c_gpu_info;
  int *d_gpu_info;
  int *gpu_info;
  int *gpu_info_t;
  bool validLaunchParams;
  clock_t start, end;

  start = clock();
  
  if (!isInitialized_native_multi_target_detection_ldl) {
    native_multi_target_detection_ldl_initialize();
  }
  
  end = clock();
  hipDeviceSynchronize();
  printf("Initialize: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();


  checkCudaError(mwCudaMalloc(&d_gpu_info, 4UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&c_gpu_info, 4UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&b_gpu_IPIV, 60UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&b_gpu_W, 180UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&b_gpu_A, 900UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_t2, 900UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&b_gpu_a, 10140UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&b_gpu_info, 4UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_info, 4UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_IPIV, 676UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_W, 10140UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_A, 114244UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_info_t, 4UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_R, 114244UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_a, 2768896UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&c_gpu_W, 2028UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_X, 2768896UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_C, 180UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_T, 10140UL), __FILE__, __LINE__);
  //  R = correlation_matrix(X);
  //  TIMING_0
  
  end = clock();
  hipDeviceSynchronize();
  printf("Allocate: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();
  
  checkCudaError(hipMemcpy(*gpu_X, cpu_X, 2768896UL, hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
                 
  end = clock();
  hipDeviceSynchronize();
  printf("Copy: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();
  
  native_multi_target_detection_ldl_kernel1<<<dim3(1352U, 1U, 1U),
                                              dim3(512U, 1U, 1U)>>>(*gpu_X,
                                                                    *gpu_a);
  alpha1 = 1.0F;
  beta1 = 0.0F;
  cublasCheck(hipblasSgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                          169, 169, 4096, (float *)&alpha1,
                          (float *)&(*gpu_a)[0], 169, (float *)&(*gpu_X)[0],
                          4096, (float *)&beta1, (float *)&(*gpu_R)[0], 169),
              __FILE__, __LINE__);
  //  TIMING_1
  end = clock();
  hipDeviceSynchronize();
  printf("Correlation matrix 0: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();
  
  native_multi_target_detection_ldl_kernel2<<<dim3(100U, 1U, 1U),
                                              dim3(288U, 1U, 1U)>>>(*gpu_R);
  //  TIMING_2
  //  Alternative: t1 = R \ T;
  // t1 = R \ T;
  end = clock();
  hipDeviceSynchronize();
  printf("Correlation matrix 1: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();


  cusolverCheck(hipsolverDnSpotrf_bufferSize(
                    getCuSolverGlobalHandle(), HIPBLAS_FILL_MODE_UPPER, 169,
                    (float *)&(*gpu_R)[0], 169, getCuSolverWorkspaceReq()),
                __FILE__, __LINE__);
  setCuSolverWorkspaceTypeSize(4);
  cusolverInitWorkspace();
  cusolverCheck(
      hipsolverDnSpotrf(getCuSolverGlobalHandle(), HIPBLAS_FILL_MODE_UPPER, 169,
                       (float *)&(*gpu_R)[0], 169,
                       static_cast<float *>(getCuSolverWorkspaceBuff()),
                       *getCuSolverWorkspaceReq(), gpu_info_t),
      __FILE__, __LINE__);
  checkCudaError(
      hipMemcpy(&cpu_info_t, gpu_info_t, 4UL, hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  if (cpu_info_t < 0) {
    native_multi_target_detection_ldl_kernel3<<<dim3(100U, 1U, 1U),
                                                dim3(288U, 1U, 1U)>>>(*gpu_R);
  }
  if (cpu_info_t == 0) {
    cpu_info_t = 169;
  } else {
    cpu_info_t--;
  }
  mwGetLaunchParameters1D(computeNumIters(cpu_info_t - 2), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    native_multi_target_detection_ldl_kernel4<<<grid, block>>>(
        cpu_info_t, cpu_info_t - 2, *gpu_R);
  }
  native_multi_target_detection_ldl_kernel5<<<dim3(100U, 1U, 1U),
                                              dim3(288U, 1U, 1U)>>>(*gpu_R,
                                                                    *gpu_A);
  checkCudaError(hipMemcpy(*gpu_T, cpu_T, 10140UL, hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
  native_multi_target_detection_ldl_kernel6<<<dim3(20U, 1U, 1U),
                                              dim3(128U, 1U, 1U)>>>(*gpu_T,
                                                                    *gpu_W);
  cusolverCheck(hipsolverDnSgetrf_bufferSize(getCuSolverGlobalHandle(), 169, 169,
                                            (float *)&(*gpu_A)[0], 169,
                                            getCuSolverWorkspaceReq()),
                __FILE__, __LINE__);
  setCuSolverWorkspaceTypeSize(4);
  cusolverInitWorkspace();
  cusolverCheck(hipsolverDnSgetrf(
                    getCuSolverGlobalHandle(), 169, 169, (float *)&(*gpu_A)[0],
                    169, static_cast<float *>(getCuSolverWorkspaceBuff()),
                    &(*gpu_IPIV)[0], gpu_info),
                __FILE__, __LINE__);
  checkCudaError(hipMemcpy(&cpu_info, gpu_info, 4UL, hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
  if (cpu_info < 0) {
    native_multi_target_detection_ldl_kernel7<<<dim3(20U, 1U, 1U),
                                                dim3(128U, 1U, 1U)>>>(*gpu_W);
  } else {
    cusolverCheck(hipsolverDnSgetrs(getCuSolverGlobalHandle(), HIPBLAS_OP_N, 169,
                                   15, (float *)&(*gpu_A)[0], 169,
                                   &(*gpu_IPIV)[0], (float *)&(*gpu_W)[0], 169,
                                   gpu_info_t),
                  __FILE__, __LINE__);
  }
  cusolverCheck(hipsolverDnSgetrf_bufferSize(getCuSolverGlobalHandle(), 169, 169,
                                            (float *)&(*gpu_R)[0], 169,
                                            getCuSolverWorkspaceReq()),
                __FILE__, __LINE__);
  setCuSolverWorkspaceTypeSize(4);
  cusolverInitWorkspace();
  cusolverCheck(hipsolverDnSgetrf(
                    getCuSolverGlobalHandle(), 169, 169, (float *)&(*gpu_R)[0],
                    169, static_cast<float *>(getCuSolverWorkspaceBuff()),
                    &(*gpu_IPIV)[0], b_gpu_info),
                __FILE__, __LINE__);
  checkCudaError(
      hipMemcpy(&b_cpu_info, b_gpu_info, 4UL, hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  if (b_cpu_info < 0) {
    native_multi_target_detection_ldl_kernel8<<<dim3(20U, 1U, 1U),
                                                dim3(128U, 1U, 1U)>>>(*gpu_W);
  } else {
    cusolverCheck(hipsolverDnSgetrs(getCuSolverGlobalHandle(), HIPBLAS_OP_N, 169,
                                   15, (float *)&(*gpu_R)[0], 169,
                                   &(*gpu_IPIV)[0], (float *)&(*gpu_W)[0], 169,
                                   gpu_info_t),
                  __FILE__, __LINE__);
  }
  //  TIMING_3
    end = clock();
  hipDeviceSynchronize();
  printf("LDL 1: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();


  native_multi_target_detection_ldl_kernel9<<<dim3(20U, 1U, 1U),
                                              dim3(128U, 1U, 1U)>>>(*gpu_T,
                                                                    *b_gpu_a);
  cublasCheck(hipblasSgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, 15,
                          15, 169, (float *)&alpha1, (float *)&(*b_gpu_a)[0],
                          15, (float *)&(*gpu_W)[0], 169, (float *)&beta1,
                          (float *)&(*gpu_t2)[0], 15),
              __FILE__, __LINE__);
  //  TIMING_4
  //  Alternative: t3 = t2 \ C;
  // t3 = t2 \ C;
    end = clock();
  hipDeviceSynchronize();
  printf("MUL 1: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();
  
  cusolverCheck(hipsolverDnSpotrf_bufferSize(
                    getCuSolverGlobalHandle(), HIPBLAS_FILL_MODE_UPPER, 15,
                    (float *)&(*gpu_t2)[0], 15, getCuSolverWorkspaceReq()),
                __FILE__, __LINE__);
  setCuSolverWorkspaceTypeSize(4);
  cusolverInitWorkspace();
  cusolverCheck(
      hipsolverDnSpotrf(getCuSolverGlobalHandle(), HIPBLAS_FILL_MODE_UPPER, 15,
                       (float *)&(*gpu_t2)[0], 15,
                       static_cast<float *>(getCuSolverWorkspaceBuff()),
                       *getCuSolverWorkspaceReq(), gpu_info_t),
      __FILE__, __LINE__);
  checkCudaError(
      hipMemcpy(&cpu_info_t, gpu_info_t, 4UL, hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  if (cpu_info_t < 0) {
    native_multi_target_detection_ldl_kernel10<<<dim3(2U, 1U, 1U),
                                                 dim3(128U, 1U, 1U)>>>(*gpu_t2);
  }
  if (cpu_info_t == 0) {
    cpu_info_t = 15;
  } else {
    cpu_info_t--;
  }
  mwGetLaunchParameters1D(computeNumIters(cpu_info_t - 2), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    native_multi_target_detection_ldl_kernel11<<<grid, block>>>(
        cpu_info_t, cpu_info_t - 2, *gpu_t2);
  }
  native_multi_target_detection_ldl_kernel12<<<dim3(2U, 1U, 1U),
                                               dim3(128U, 1U, 1U)>>>(*gpu_t2,
                                                                     *b_gpu_A);
  checkCudaError(hipMemcpy(*gpu_C, cpu_C, 180UL, hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
  native_multi_target_detection_ldl_kernel13<<<dim3(1U, 1U, 1U),
                                               dim3(64U, 1U, 1U)>>>(*gpu_C,
                                                                    *b_gpu_W);
  cusolverCheck(hipsolverDnSgetrf_bufferSize(getCuSolverGlobalHandle(), 15, 15,
                                            (float *)&(*b_gpu_A)[0], 15,
                                            getCuSolverWorkspaceReq()),
                __FILE__, __LINE__);
  setCuSolverWorkspaceTypeSize(4);
  cusolverInitWorkspace();
  cusolverCheck(hipsolverDnSgetrf(
                    getCuSolverGlobalHandle(), 15, 15, (float *)&(*b_gpu_A)[0],
                    15, static_cast<float *>(getCuSolverWorkspaceBuff()),
                    &(*b_gpu_IPIV)[0], c_gpu_info),
                __FILE__, __LINE__);
  checkCudaError(
      hipMemcpy(&c_cpu_info, c_gpu_info, 4UL, hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  if (c_cpu_info < 0) {
    native_multi_target_detection_ldl_kernel14<<<dim3(1U, 1U, 1U),
                                                 dim3(64U, 1U, 1U)>>>(*b_gpu_W);
  } else {
    cusolverCheck(hipsolverDnSgetrs(getCuSolverGlobalHandle(), HIPBLAS_OP_N, 15,
                                   3, (float *)&(*b_gpu_A)[0], 15,
                                   &(*b_gpu_IPIV)[0], (float *)&(*b_gpu_W)[0],
                                   15, gpu_info_t),
                  __FILE__, __LINE__);
  }
  cusolverCheck(hipsolverDnSgetrf_bufferSize(getCuSolverGlobalHandle(), 15, 15,
                                            (float *)&(*gpu_t2)[0], 15,
                                            getCuSolverWorkspaceReq()),
                __FILE__, __LINE__);
  setCuSolverWorkspaceTypeSize(4);
  cusolverInitWorkspace();
  cusolverCheck(hipsolverDnSgetrf(
                    getCuSolverGlobalHandle(), 15, 15, (float *)&(*gpu_t2)[0],
                    15, static_cast<float *>(getCuSolverWorkspaceBuff()),
                    &(*b_gpu_IPIV)[0], d_gpu_info),
                __FILE__, __LINE__);
  checkCudaError(
      hipMemcpy(&d_cpu_info, d_gpu_info, 4UL, hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  if (d_cpu_info < 0) {
    native_multi_target_detection_ldl_kernel15<<<dim3(1U, 1U, 1U),
                                                 dim3(64U, 1U, 1U)>>>(*b_gpu_W);
  } else {
    cusolverCheck(hipsolverDnSgetrs(getCuSolverGlobalHandle(), HIPBLAS_OP_N, 15,
                                   3, (float *)&(*gpu_t2)[0], 15,
                                   &(*b_gpu_IPIV)[0], (float *)&(*b_gpu_W)[0],
                                   15, gpu_info_t),
                  __FILE__, __LINE__);
  }
  //  TIMING_5
    end = clock();
  hipDeviceSynchronize();
  printf("LDL 2: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();
  
  //  Compute the weighting matrix W
  cublasCheck(hipblasSgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                          169, 3, 15, (float *)&alpha1, (float *)&(*gpu_W)[0],
                          169, (float *)&(*b_gpu_W)[0], 15, (float *)&beta1,
                          (float *)&(*c_gpu_W)[0], 169),
              __FILE__, __LINE__);
  //  TIMING_6
    end = clock();
  hipDeviceSynchronize();
  printf("MUL 2: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();
  
  
  checkCudaError(hipMemcpy(cpu_W, *c_gpu_W, 2028UL, hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
                  

  end = clock();
  hipDeviceSynchronize();
  printf("Copy back: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();


  checkCudaError(mwCudaFree(*gpu_T), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_C), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_X), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*c_gpu_W), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_a), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_R), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(gpu_info_t), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_A), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_W), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_IPIV), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(gpu_info), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(b_gpu_info), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*b_gpu_a), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_t2), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*b_gpu_A), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*b_gpu_W), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*b_gpu_IPIV), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(c_gpu_info), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(d_gpu_info), __FILE__, __LINE__);
  
    end = clock();
  hipDeviceSynchronize();
  printf("Free: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();


}

//
// File trailer for native_multi_target_detection_ldl.cu
//
// [EOF]
//

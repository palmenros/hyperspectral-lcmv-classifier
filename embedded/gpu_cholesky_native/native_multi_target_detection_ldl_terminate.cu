//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: native_multi_target_detection_ldl_terminate.cu
//
// GPU Coder version                    : 24.1
// CUDA/C/C++ source code generated on  : 17-May-2024 07:49:02
//

// Include Files
#include "native_multi_target_detection_ldl_terminate.h"
#include "native_multi_target_detection_ldl_data.h"
#include "native_multi_target_detection_ldl_rtwutil.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"
#include "MWMemoryManager.hpp"
#include "stdio.h"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void native_multi_target_detection_ldl_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "ERR[%d] %s:%s\n", errCode, hipGetErrorName(errCode),
            hipGetErrorString(errCode));
    exit(errCode);
  }
  cublasEnsureDestruction();
  cusolverDestroyWorkspace();
  cusolverEnsureDestruction();
  checkCudaError(mwMemoryManagerTerminate(), __FILE__, __LINE__);
  isInitialized_native_multi_target_detection_ldl = false;
}

//
// File trailer for native_multi_target_detection_ldl_terminate.cu
//
// [EOF]
//

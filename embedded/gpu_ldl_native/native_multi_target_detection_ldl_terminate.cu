//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: native_multi_target_detection_ldl_terminate.cu
//
// GPU Coder version                    : 24.1
// CUDA/C/C++ source code generated on  : 12-May-2024 11:24:45
//

// Include Files
#include "native_multi_target_detection_ldl_terminate.h"
#include "native_multi_target_detection_ldl_data.h"
#include "native_multi_target_detection_ldl_rtwutil.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"
#include "MWMemoryManager.hpp"
#include "stdio.h"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void native_multi_target_detection_ldl_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "ERR[%d] %s:%s\n", errCode, hipGetErrorName(errCode),
            hipGetErrorString(errCode));
    exit(errCode);
  }
  cublasEnsureDestruction();
  cusolverDestroyWorkspace();
  cusolverEnsureDestruction();
  checkCudaError(mwMemoryManagerTerminate(), __FILE__, __LINE__);
  isInitialized_native_multi_target_detection_ldl = false;
}

//
// File trailer for native_multi_target_detection_ldl_terminate.cu
//
// [EOF]
//

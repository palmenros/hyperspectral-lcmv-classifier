//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: native_multi_target_detection_ldl_initialize.cu
//
// GPU Coder version                    : 24.1
// CUDA/C/C++ source code generated on  : 12-May-2024 11:24:45
//

// Include Files
#include "native_multi_target_detection_ldl_initialize.h"
#include "native_multi_target_detection_ldl_data.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"
#include "MWMemoryManager.hpp"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void native_multi_target_detection_ldl_initialize()
{
  hipGetLastError();
  mwMemoryManagerInit(256U, 0U, 8U, 2048U);
  cublasEnsureInitialization(HIPBLAS_POINTER_MODE_HOST);
  cusolverEnsureInitialization();
  isInitialized_native_multi_target_detection_ldl = true;
}

//
// File trailer for native_multi_target_detection_ldl_initialize.cu
//
// [EOF]
//

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: native_multi_target_detection_ldl.cu
//
// GPU Coder version                    : 24.1
// CUDA/C/C++ source code generated on  : 12-May-2024 11:24:45
//

// Include Files
#include "native_multi_target_detection_ldl.h"
#include "native_multi_target_detection_ldl_data.h"
#include "native_multi_target_detection_ldl_initialize.h"
#include "native_multi_target_detection_ldl_rtwutil.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWErrorCodeUtils.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "hip/hip_math_constants.h"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include <cmath>
#include <time.h>

// Function Declarations
static long computeEndIdx(long start, long end, long stride);

static
#ifdef __HIPCC__
    __device__
#endif
    long
    computeEndIdx_device(long start, long end, long stride);

static unsigned long computeNumIters(long ub);

static void cublasCheck(hipblasStatus_t errCode, const char *file,
                        unsigned int b_line);

static void cusolverCheck(hipsolverStatus_t errCode, const char *file,
                          unsigned int b_line);

static __global__ void
native_multi_target_detection_ldl_kernel1(const float X[692224],
                                          float a[692224]);

static __global__ void
native_multi_target_detection_ldl_kernel10(const float T[2535], float a[2535]);

static __global__ void native_multi_target_detection_ldl_kernel11(float D[225],
                                                                  float L[225]);

static __global__ void
native_multi_target_detection_ldl_kernel12(const float C[45], float W[45]);

static __global__ void
native_multi_target_detection_ldl_kernel13(const float L[225], float t2[225]);

static __global__ void native_multi_target_detection_ldl_kernel14(float W[45]);

static __global__ void native_multi_target_detection_ldl_kernel15(float W[45]);

static __global__ void
native_multi_target_detection_ldl_kernel16(const float L[225], float t2[225]);

static __global__ void native_multi_target_detection_ldl_kernel17(float W[45]);

static __global__ void
native_multi_target_detection_ldl_kernel18(const float d11, const int ix,
                                           const long b, float t2[225]);

static __global__ void native_multi_target_detection_ldl_kernel19(float L[225]);

static __global__ void
native_multi_target_detection_ldl_kernel2(float R[28561]);

static __global__ void native_multi_target_detection_ldl_kernel20(const int k,
                                                                  float L[225]);

static __global__ void native_multi_target_detection_ldl_kernel21(const int k,
                                                                  float L[225]);

static __global__ void
native_multi_target_detection_ldl_kernel22(const float t2[225], const int k,
                                           const long b, float L[225]);

static __global__ void native_multi_target_detection_ldl_kernel23(float D[225]);

static __global__ void
native_multi_target_detection_ldl_kernel24(const float d11, const int ix,
                                           const long b, float R[28561]);

static __global__ void
native_multi_target_detection_ldl_kernel25(float L[28561]);

static __global__ void
native_multi_target_detection_ldl_kernel26(const int k, float L[28561]);

static __global__ void
native_multi_target_detection_ldl_kernel27(const int k, float L[28561]);

static __global__ void
native_multi_target_detection_ldl_kernel28(const float R[28561], const int k,
                                           const long b, float L[28561]);

static __global__ void
native_multi_target_detection_ldl_kernel29(float D[28561]);

static __global__ void
native_multi_target_detection_ldl_kernel3(float D[28561], float L[28561]);

static __global__ void
native_multi_target_detection_ldl_kernel4(const float T[2535], float W[2535]);

static __global__ void
native_multi_target_detection_ldl_kernel5(const float L[28561], float R[28561]);

static __global__ void native_multi_target_detection_ldl_kernel6(float W[2535]);

static __global__ void native_multi_target_detection_ldl_kernel7(float W[2535]);

static __global__ void
native_multi_target_detection_ldl_kernel8(const float L[28561], float R[28561]);

static __global__ void native_multi_target_detection_ldl_kernel9(float W[2535]);

static void raiseCudaError(int errCode, const char *file, unsigned int b_line,
                           const char *errorName, const char *errorString);

// Function Definitions
//
// Arguments    : long start
//                long end
//                long stride
// Return Type  : long
//
static long computeEndIdx(long start, long end, long stride)
{
  long newEnd;
  newEnd = -1L;
  if ((stride > 0L) && (start <= end)) {
    newEnd = (end - start) / stride;
  } else if ((stride < 0L) && (end <= start)) {
    newEnd = (start - end) / -stride;
  }
  return newEnd;
}

//
// Arguments    : long start
//                long end
//                long stride
// Return Type  : long
//
static __device__ long computeEndIdx_device(long start, long end, long stride)
{
  long newEnd;
  newEnd = -1L;
  if ((stride > 0L) && (start <= end)) {
    newEnd = (end - start) / stride;
  } else if ((stride < 0L) && (end <= start)) {
    newEnd = (start - end) / -stride;
  }
  return newEnd;
}

//
// Arguments    : long ub
// Return Type  : unsigned long
//
static unsigned long computeNumIters(long ub)
{
  unsigned long numIters;
  numIters = 0UL;
  if (ub >= 0L) {
    numIters = static_cast<unsigned long>(ub + 1L);
  }
  return numIters;
}

//
// Arguments    : hipblasStatus_t errCode
//                const char *file
//                unsigned int b_line
// Return Type  : void
//
static void cublasCheck(hipblasStatus_t errCode, const char *file,
                        unsigned int b_line)
{
  const char *errName;
  const char *errString;
  if (errCode != HIPBLAS_STATUS_SUCCESS) {
    cublasGetErrorName(errCode, &errName);
    cublasGetErrorString(errCode, &errString);
    raiseCudaError(errCode, file, b_line, errName, errString);
  }
}

//
// Arguments    : hipsolverStatus_t errCode
//                const char *file
//                unsigned int b_line
// Return Type  : void
//
static void cusolverCheck(hipsolverStatus_t errCode, const char *file,
                          unsigned int b_line)
{
  const char *errName;
  const char *errString;
  if (errCode != HIPSOLVER_STATUS_SUCCESS) {
    cusolverGetErrorName(errCode, &errName);
    cusolverGetErrorString(errCode, &errString);
    raiseCudaError(errCode, file, b_line, errName, errString);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float X[692224]
//                float a[692224]
// Return Type  : void
//
static __global__
    __launch_bounds__(512, 1) void native_multi_target_detection_ldl_kernel1(
        const float X[692224], float a[692224])
{
  unsigned long gThreadId;
  int i;
  int jj;
  gThreadId = mwGetGlobalThreadIndex();
  jj = static_cast<int>(gThreadId % 169UL);
  i = static_cast<int>((gThreadId - static_cast<unsigned long>(jj)) / 169UL);
  if ((i < 4096) && (jj < 169)) {
    //  R = correlation_matrix(X);
    //  TIMING_0
    a[jj + 169 * i] = X[i + (jj << 12)];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float T[2535]
//                float a[2535]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void native_multi_target_detection_ldl_kernel10(const float T[2535],
                                                            float a[2535])
{
  unsigned long gThreadId;
  int i;
  int jj;
  gThreadId = mwGetGlobalThreadIndex();
  jj = static_cast<int>(gThreadId % 15UL);
  i = static_cast<int>((gThreadId - static_cast<unsigned long>(jj)) / 15UL);
  if ((i < 169) && (jj < 15)) {
    //  TIMING_3
    a[jj + 15 * i] = T[i + 169 * jj];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float D[225]
//                float L[225]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void native_multi_target_detection_ldl_kernel11(float D[225],
                                                            float L[225])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 225) {
    L[i] = HIP_NAN_F;
    D[i] = HIP_NAN_F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float C[45]
//                float W[45]
// Return Type  : void
//
static __global__ __launch_bounds__(
    64, 1) void native_multi_target_detection_ldl_kernel12(const float C[45],
                                                           float W[45])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 45) {
    W[i] = C[i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float L[225]
//                float t2[225]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void native_multi_target_detection_ldl_kernel13(const float L[225],
                                                            float t2[225])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 225) {
    t2[i] = L[i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float W[45]
// Return Type  : void
//
static __global__ __launch_bounds__(
    64, 1) void native_multi_target_detection_ldl_kernel14(float W[45])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 45) {
    W[i] = HIP_NAN_F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float W[45]
// Return Type  : void
//
static __global__ __launch_bounds__(
    64, 1) void native_multi_target_detection_ldl_kernel15(float W[45])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 45) {
    W[i] = HIP_NAN_F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float L[225]
//                float t2[225]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void native_multi_target_detection_ldl_kernel16(const float L[225],
                                                            float t2[225])
{
  unsigned long gThreadId;
  int i;
  int jj;
  gThreadId = mwGetGlobalThreadIndex();
  jj = static_cast<int>(gThreadId % 15UL);
  i = static_cast<int>((gThreadId - static_cast<unsigned long>(jj)) / 15UL);
  if ((i < 15) && (jj < 15)) {
    t2[jj + 15 * i] = L[i + 15 * jj];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float W[45]
// Return Type  : void
//
static __global__ __launch_bounds__(
    64, 1) void native_multi_target_detection_ldl_kernel17(float W[45])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 45) {
    W[i] = HIP_NAN_F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float d11
//                const int ix
//                const long b
//                float t2[225]
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void native_multi_target_detection_ldl_kernel18(
        const float d11, const int ix, const long b, float t2[225])
{
  unsigned long gStride;
  unsigned long gThreadId;
  unsigned long loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(b);
  for (unsigned long idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    long jj;
    jj = static_cast<long>(idx);
    t2[static_cast<int>(static_cast<long>(ix) + jj) - 1] *= d11;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float L[225]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void native_multi_target_detection_ldl_kernel19(float L[225])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 225) {
    L[i] = 0.0F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float R[28561]
// Return Type  : void
//
static __global__ __launch_bounds__(
    288, 1) void native_multi_target_detection_ldl_kernel2(float R[28561])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 28561) {
    //  TIMING_1
    R[i] /= 4096.0F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int k
//                float L[225]
// Return Type  : void
//
static __global__ __launch_bounds__(
    32, 1) void native_multi_target_detection_ldl_kernel20(const int k,
                                                           float L[225])
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    L[(k + 15 * (k - 1)) - 1] = 1.0F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int k
//                float L[225]
// Return Type  : void
//
static __global__ __launch_bounds__(
    32, 1) void native_multi_target_detection_ldl_kernel21(const int k,
                                                           float L[225])
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    L[k + 15 * k] = 1.0F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float t2[225]
//                const int k
//                const long b
//                float L[225]
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void native_multi_target_detection_ldl_kernel22(
        const float t2[225], const int k, const long b, float L[225])
{
  unsigned long gStride;
  unsigned long gThreadId;
  unsigned long loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(b);
  for (unsigned long idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    long b_jj;
    int i;
    int jj;
    b_jj = static_cast<long>(idx);
    jj = static_cast<int>(static_cast<long>(k) + b_jj);
    i = k + 2;
    for (b_jj = 0L; b_jj <= computeEndIdx_device(static_cast<long>(i), 15L, 1L);
         b_jj++) {
      L[(static_cast<int>(static_cast<long>(k + 2) + b_jj) + 15 * (jj - 1)) -
        1] = t2[(static_cast<int>(static_cast<long>(k + 2) + b_jj) +
                 15 * (jj - 1)) -
                1];
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float D[225]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void native_multi_target_detection_ldl_kernel23(float D[225])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 225) {
    D[i] = 0.0F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float d11
//                const int ix
//                const long b
//                float R[28561]
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void native_multi_target_detection_ldl_kernel24(
        const float d11, const int ix, const long b, float R[28561])
{
  unsigned long gStride;
  unsigned long gThreadId;
  unsigned long loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(b);
  for (unsigned long idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    long jj;
    jj = static_cast<long>(idx);
    R[static_cast<int>(static_cast<long>(ix) + jj) - 1] *= d11;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float L[28561]
// Return Type  : void
//
static __global__ __launch_bounds__(
    288, 1) void native_multi_target_detection_ldl_kernel25(float L[28561])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 28561) {
    L[i] = 0.0F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int k
//                float L[28561]
// Return Type  : void
//
static __global__ __launch_bounds__(
    32, 1) void native_multi_target_detection_ldl_kernel26(const int k,
                                                           float L[28561])
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    L[(k + 169 * (k - 1)) - 1] = 1.0F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int k
//                float L[28561]
// Return Type  : void
//
static __global__ __launch_bounds__(
    32, 1) void native_multi_target_detection_ldl_kernel27(const int k,
                                                           float L[28561])
{
  int tmpIdx;
  tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    L[k + 169 * k] = 1.0F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float R[28561]
//                const int k
//                const long b
//                float L[28561]
// Return Type  : void
//
static __global__
    __launch_bounds__(1024, 1) void native_multi_target_detection_ldl_kernel28(
        const float R[28561], const int k, const long b, float L[28561])
{
  unsigned long gStride;
  unsigned long gThreadId;
  unsigned long loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(b);
  for (unsigned long idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    long b_jj;
    int i;
    int jj;
    b_jj = static_cast<long>(idx);
    jj = static_cast<int>(static_cast<long>(k) + b_jj);
    i = k + 2;
    for (b_jj = 0L;
         b_jj <= computeEndIdx_device(static_cast<long>(i), 169L, 1L); b_jj++) {
      L[(static_cast<int>(static_cast<long>(k + 2) + b_jj) + 169 * (jj - 1)) -
        1] = R[(static_cast<int>(static_cast<long>(k + 2) + b_jj) +
                169 * (jj - 1)) -
               1];
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float D[28561]
// Return Type  : void
//
static __global__ __launch_bounds__(
    288, 1) void native_multi_target_detection_ldl_kernel29(float D[28561])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 28561) {
    D[i] = 0.0F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float D[28561]
//                float L[28561]
// Return Type  : void
//
static __global__ __launch_bounds__(
    288, 1) void native_multi_target_detection_ldl_kernel3(float D[28561],
                                                           float L[28561])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 28561) {
    L[i] = HIP_NAN_F;
    D[i] = HIP_NAN_F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float T[2535]
//                float W[2535]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void native_multi_target_detection_ldl_kernel4(const float T[2535],
                                                           float W[2535])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 2535) {
    W[i] = T[i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float L[28561]
//                float R[28561]
// Return Type  : void
//
static __global__ __launch_bounds__(
    288, 1) void native_multi_target_detection_ldl_kernel5(const float L[28561],
                                                           float R[28561])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 28561) {
    R[i] = L[i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float W[2535]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void native_multi_target_detection_ldl_kernel6(float W[2535])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 2535) {
    W[i] = HIP_NAN_F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float W[2535]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void native_multi_target_detection_ldl_kernel7(float W[2535])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 2535) {
    W[i] = HIP_NAN_F;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float L[28561]
//                float R[28561]
// Return Type  : void
//
static __global__ __launch_bounds__(
    288, 1) void native_multi_target_detection_ldl_kernel8(const float L[28561],
                                                           float R[28561])
{
  unsigned long gThreadId;
  int i;
  int jj;
  gThreadId = mwGetGlobalThreadIndex();
  jj = static_cast<int>(gThreadId % 169UL);
  i = static_cast<int>((gThreadId - static_cast<unsigned long>(jj)) / 169UL);
  if ((i < 169) && (jj < 169)) {
    R[jj + 169 * i] = L[i + 169 * jj];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                float W[2535]
// Return Type  : void
//
static __global__ __launch_bounds__(
    128, 1) void native_multi_target_detection_ldl_kernel9(float W[2535])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 2535) {
    W[i] = HIP_NAN_F;
  }
}

//
// Arguments    : int errCode
//                const char *file
//                unsigned int b_line
//                const char *errorName
//                const char *errorString
// Return Type  : void
//
static void raiseCudaError(int errCode, const char *file, unsigned int b_line,
                           const char *errorName, const char *errorString)
{
  printf("ERR[%d] %s:%s in file %s at line %d\nExiting program execution ...\n",
         errCode, errorName, errorString, file, b_line);
  exit(errCode);
}

//
// Arguments    : const float cpu_T[2535]
//                const float cpu_C[45]
//                const float cpu_X[692224]
//                float cpu_W[507]
// Return Type  : void
//
void native_multi_target_detection_ldl(const float cpu_T[2535],
                                       const float cpu_C[45],
                                       const float cpu_X[692224],
                                       float cpu_W[507])
{
  static float cpu_L[28561];
  static float cpu_R[28561];
  dim3 block;
  dim3 grid;
  long b;
  long c;
  float(*gpu_X)[692224];
  float(*gpu_a)[692224];
  float cpu_D[28561];
  float(*gpu_D)[28561];
  float(*gpu_L)[28561];
  float(*gpu_R)[28561];
  float(*b_gpu_a)[2535];
  float(*gpu_T)[2535];
  float(*gpu_W)[2535];
  float(*c_gpu_W)[507];
  float b_cpu_D[225];
  float b_cpu_L[225];
  float cpu_t2[225];
  float(*b_gpu_D)[225];
  float(*b_gpu_L)[225];
  float(*gpu_t2)[225];
  float(*b_gpu_W)[45];
  float(*gpu_C)[45];
  float colmax;
  float d11;
  float s;
  float smax;
  float temp;
  float wkp1;
  int(*gpu_IPIV)[169];
  int(*b_gpu_IPIV)[15];
  int b_cpu_info;
  int b_p;
  int c_cpu_info;
  int cpu_info;
  int cpu_iy;
  int d_cpu_info;
  int e_cpu_info;
  int exitg1;
  int f_cpu_info;
  int imax;
  int ix;
  int jmax;
  int k;
  int kstep;
  int n;
  int *b_gpu_info;
  int *c_gpu_info;
  int *d_gpu_info;
  int *e_gpu_info;
  int *f_gpu_info;
  int *gpu_info;
  int *gpu_iy;
  bool L_outdatedOnCpu;
  bool L_outdatedOnGpu;
  bool R_outdatedOnCpu;
  bool R_outdatedOnGpu;
  bool p;
  bool validLaunchParams;
  clock_t start, end;
  start = clock();


  if (!isInitialized_native_multi_target_detection_ldl) {
    native_multi_target_detection_ldl_initialize();
  }
  end = clock();
  hipDeviceSynchronize();
  printf("Initialize: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();

  checkCudaError(mwCudaMalloc(&f_gpu_info, 4UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&e_gpu_info, 4UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&d_gpu_info, 4UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&b_gpu_IPIV, 60UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&b_gpu_W, 180UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&b_gpu_D, 900UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&b_gpu_L, 900UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_t2, 900UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&b_gpu_a, 10140UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&c_gpu_info, 4UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&b_gpu_info, 4UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_info, 4UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_iy, 4UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_IPIV, 676UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_W, 10140UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_D, 114244UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_L, 114244UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_R, 114244UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_a, 2768896UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&c_gpu_W, 2028UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_X, 2768896UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_C, 180UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_T, 10140UL), __FILE__, __LINE__);
  //  R = correlation_matrix(X);
  end = clock();
  hipDeviceSynchronize();
  printf("Allocate: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();


  checkCudaError(hipMemcpy(*gpu_X, cpu_X, 2768896UL, hipMemcpyHostToDevice),
                 __FILE__, __LINE__);

  end = clock();
  hipDeviceSynchronize();
  printf("Copy: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();

  //  TIMING_0
  native_multi_target_detection_ldl_kernel1<<<dim3(1352U, 1U, 1U),
                                              dim3(512U, 1U, 1U)>>>(*gpu_X,
                                                                    *gpu_a);
  temp = 1.0F;
  smax = 0.0F;
  cublasCheck(hipblasSgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                          169, 169, 4096, (float *)&temp, (float *)&(*gpu_a)[0],
                          169, (float *)&(*gpu_X)[0], 4096, (float *)&smax,
                          (float *)&(*gpu_R)[0], 169),
              __FILE__, __LINE__);
  //  TIMING_1

  end = clock();
  hipDeviceSynchronize();
  printf("Correlation matrix (0): %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();


  native_multi_target_detection_ldl_kernel2<<<dim3(100U, 1U, 1U),
                                              dim3(288U, 1U, 1U)>>>(*gpu_R);
  R_outdatedOnGpu = false;
  R_outdatedOnCpu = true;
  //  TIMING_2

  end = clock();
  hipDeviceSynchronize();
  printf("Correlation matrix (1): %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();



  //  Alternative: t1 = R \ T;
  p = true;
  for (k = 0; k < 28561; k++) {
    if (p) {
      if (R_outdatedOnCpu) {
        checkCudaError(
            hipMemcpy(cpu_R, *gpu_R, 114244UL, hipMemcpyDeviceToHost),
            __FILE__, __LINE__);
      }
      R_outdatedOnCpu = false;
      temp = cpu_R[k];
      if (std::isinf(temp) || std::isnan(temp)) {
        p = false;
      }
    } else {
      p = false;
    }
  }
  if (!p) {
    native_multi_target_detection_ldl_kernel3<<<dim3(100U, 1U, 1U),
                                                dim3(288U, 1U, 1U)>>>(*gpu_D,
                                                                      *gpu_L);
    L_outdatedOnGpu = false;
    R_outdatedOnGpu = false;
  } else {
    int ipiv[169];
    for (k = 0; k + 1 <= 169; k += kstep) {
      kstep = 1;
      b_p = k;
      if (R_outdatedOnCpu) {
        checkCudaError(
            hipMemcpy(cpu_R, *gpu_R, 114244UL, hipMemcpyDeviceToHost),
            __FILE__, __LINE__);
      }
      R_outdatedOnCpu = false;
      temp = std::abs(cpu_R[k + 169 * k]);
      if (k + 1 < 169) {
        cpu_iy = (k * 169 + k) + 1;
        n = 166 - k;
        ix = 1;
        if (168 - k > 1) {
          smax = std::abs(cpu_R[cpu_iy]);
          for (int b_k{0}; b_k <= n; b_k++) {
            s = std::abs(cpu_R[(cpu_iy + b_k) + 1]);
            if (s > smax) {
              ix = b_k + 2;
              smax = s;
            }
          }
        }
        imax = k + ix;
        colmax = std::abs(cpu_R[imax + 169 * k]);
      } else {
        imax = 168;
        colmax = 0.0F;
      }
      if (std::fmax(temp, colmax) == 0.0F) {
        imax = k;
        ipiv[k] = imax + 1;
      } else {
        if (!(temp < 0.640388191F * colmax)) {
          imax = k;
        } else {
          do {
            exitg1 = 0;
            if (imax + 1 != k + 1) {
              cpu_iy = k * 169 + imax;
              n = imax - k;
              if (n < 1) {
                ix = 0;
              } else {
                ix = 1;
                if (n > 1) {
                  smax = std::abs(cpu_R[cpu_iy]);
                  for (int b_k{0}; b_k <= n - 2; b_k++) {
                    s = std::abs(cpu_R[cpu_iy + (b_k + 1) * 169]);
                    if (s > smax) {
                      ix = b_k + 2;
                      smax = s;
                    }
                  }
                }
              }
              jmax = k + ix;
              temp = std::abs(cpu_R[imax + 169 * (jmax - 1)]);
            } else {
              temp = 0.0F;
              jmax = k;
            }
            if (imax + 1 < 169) {
              cpu_iy = (imax * 169 + imax) + 1;
              n = 166 - imax;
              ix = 2;
              if (168 - imax > 1) {
                smax = std::abs(cpu_R[cpu_iy]);
                for (int b_k{0}; b_k <= n; b_k++) {
                  s = std::abs(cpu_R[(cpu_iy + b_k) + 1]);
                  if (s > smax) {
                    ix = b_k + 3;
                    smax = s;
                  }
                }
              }
              cpu_iy = imax + ix;
              smax = std::abs(cpu_R[(cpu_iy + 169 * imax) - 1]);
              if (smax > temp) {
                temp = smax;
                jmax = cpu_iy;
              }
            }
            if (!(std::abs(cpu_R[imax + 169 * imax]) < 0.640388191F * temp)) {
              exitg1 = 1;
            } else if ((b_p + 1 == jmax) || (temp <= colmax)) {
              kstep = 2;
              exitg1 = 1;
            } else {
              b_p = imax;
              colmax = temp;
              imax = jmax - 1;
            }
          } while (exitg1 == 0);
        }
        jmax = (k + kstep) - 1;
        if ((kstep == 2) && (b_p + 1 != k + 1)) {
          if (b_p + 1 < 169) {
            ix = (k * 169 + b_p) + 1;
            cpu_iy = (b_p * 169 + b_p) + 1;
            n = 167 - b_p;
            for (int b_k{0}; b_k <= n; b_k++) {
              temp = cpu_R[ix + b_k];
              cpu_R[ix + b_k] = cpu_R[cpu_iy + b_k];
              cpu_R[cpu_iy + b_k] = temp;
            }
          }
          b = computeEndIdx(static_cast<long>(k + 2), static_cast<long>(b_p),
                            1L);
          for (long ii{0L}; ii <= b; ii++) {
            temp = cpu_R[(static_cast<int>((k + 2) + ii) + 169 * k) - 1];
            cpu_R[(static_cast<int>((k + 2) + ii) + 169 * k) - 1] =
                cpu_R[b_p + 169 * (static_cast<int>((k + 2) + ii) - 1)];
            cpu_R[b_p + 169 * (static_cast<int>((k + 2) + ii) - 1)] = temp;
          }
          temp = cpu_R[k + 169 * k];
          cpu_R[k + 169 * k] = cpu_R[b_p + 169 * b_p];
          cpu_R[b_p + 169 * b_p] = temp;
          R_outdatedOnGpu = true;
        }
        if (imax + 1 != jmax + 1) {
          if (imax + 1 < 169) {
            ix = (jmax * 169 + imax) + 1;
            cpu_iy = (imax * 169 + imax) + 1;
            n = 167 - imax;
            for (int b_k{0}; b_k <= n; b_k++) {
              temp = cpu_R[ix + b_k];
              cpu_R[ix + b_k] = cpu_R[cpu_iy + b_k];
              cpu_R[cpu_iy + b_k] = temp;
            }
          }
          b = computeEndIdx(static_cast<long>(jmax + 2),
                            static_cast<long>(imax), 1L);
          for (long ii{0L}; ii <= b; ii++) {
            temp = cpu_R[(static_cast<int>((jmax + 2) + ii) + 169 * jmax) - 1];
            cpu_R[(static_cast<int>((jmax + 2) + ii) + 169 * jmax) - 1] =
                cpu_R[imax + 169 * (static_cast<int>((jmax + 2) + ii) - 1)];
            cpu_R[imax + 169 * (static_cast<int>((jmax + 2) + ii) - 1)] = temp;
          }
          temp = cpu_R[jmax + 169 * jmax];
          cpu_R[jmax + 169 * jmax] = cpu_R[imax + 169 * imax];
          cpu_R[imax + 169 * imax] = temp;
          R_outdatedOnGpu = true;
          if (kstep == 2) {
            temp = cpu_R[(k + 169 * k) + 1];
            cpu_R[(k + 169 * k) + 1] = cpu_R[imax + 169 * k];
            cpu_R[imax + 169 * k] = temp;
          }
        }
        if (kstep == 1) {
          if (k + 1 < 169) {
            if (std::abs(cpu_R[k + 169 * k]) >= 9.09494702E-13F) {
              d11 = 1.0F / cpu_R[k + 169 * k];
              b = computeEndIdx(static_cast<long>(k + 2), 169L, 1L);
              for (long ii{0L}; ii <= b; ii++) {
                if (cpu_R[(static_cast<int>((k + 2) + ii) + 169 * k) - 1] !=
                    0.0F) {
                  temp = -d11 *
                         cpu_R[(static_cast<int>((k + 2) + ii) + 169 * k) - 1];
                  cpu_R[(static_cast<int>((k + 2) + ii) +
                         169 * (static_cast<int>((k + 2) + ii) - 1)) -
                        1] +=
                      temp *
                      cpu_R[(static_cast<int>((k + 2) + ii) + 169 * k) - 1];
                  R_outdatedOnGpu = true;
                  ix = static_cast<int>((k + 2) + ii) + 1;
                  c = computeEndIdx(static_cast<long>(ix), 169L, 1L);
                  for (long i{0L}; i <= c; i++) {
                    cpu_R[(static_cast<int>(ix + i) +
                           169 * (static_cast<int>((k + 2) + ii) - 1)) -
                          1] +=
                        cpu_R[(static_cast<int>(ix + i) + 169 * k) - 1] * temp;
                  }
                }
              }
              cpu_iy = k * 169 + k;
              b = computeEndIdx(static_cast<long>(cpu_iy + 2),
                                static_cast<long>((cpu_iy - k) + 169), 1L);
              mwGetLaunchParameters1D(computeNumIters(b), &grid, &block,
                                      2147483647U);
              if (R_outdatedOnGpu) {
                checkCudaError(
                    hipMemcpy(*gpu_R, cpu_R, 114244UL, hipMemcpyHostToDevice),
                    __FILE__, __LINE__);
              }
              validLaunchParams = mwValidateLaunchParameters(grid, block);
              if (validLaunchParams) {
                native_multi_target_detection_ldl_kernel24<<<grid, block>>>(
                    d11, cpu_iy + 2, b, *gpu_R);
              }
              R_outdatedOnGpu = false;
              R_outdatedOnCpu = true;
            } else {
              d11 = cpu_R[k + 169 * k];
              b = computeEndIdx(static_cast<long>(k + 2), 169L, 1L);
              for (long ii{0L}; ii <= b; ii++) {
                cpu_R[(static_cast<int>((k + 2) + ii) + 169 * k) - 1] /= d11;
                R_outdatedOnGpu = true;
              }
              b = computeEndIdx(static_cast<long>(k + 2), 169L, 1L);
              for (long ii{0L}; ii <= b; ii++) {
                if (cpu_R[(static_cast<int>((k + 2) + ii) + 169 * k) - 1] !=
                    0.0F) {
                  temp = -d11 *
                         cpu_R[(static_cast<int>((k + 2) + ii) + 169 * k) - 1];
                  cpu_R[(static_cast<int>((k + 2) + ii) +
                         169 * (static_cast<int>((k + 2) + ii) - 1)) -
                        1] +=
                      temp *
                      cpu_R[(static_cast<int>((k + 2) + ii) + 169 * k) - 1];
                  R_outdatedOnGpu = true;
                  ix = static_cast<int>((k + 2) + ii) + 1;
                  c = computeEndIdx(static_cast<long>(ix), 169L, 1L);
                  for (long i{0L}; i <= c; i++) {
                    cpu_R[(static_cast<int>(ix + i) +
                           169 * (static_cast<int>((k + 2) + ii) - 1)) -
                          1] +=
                        cpu_R[(static_cast<int>(ix + i) + 169 * k) - 1] * temp;
                  }
                }
              }
            }
          }
          ipiv[k] = imax + 1;
        } else {
          if (k + 1 < 168) {
            temp = cpu_R[(k + 169 * k) + 1];
            d11 = cpu_R[(k + 169 * (k + 1)) + 1] / cpu_R[(k + 169 * k) + 1];
            smax = cpu_R[k + 169 * k] / cpu_R[(k + 169 * k) + 1];
            s = 1.0F / (d11 * smax - 1.0F);
            b = computeEndIdx(static_cast<long>(k + 3), 169L, 1L);
            for (long ii{0L}; ii <= b; ii++) {
              colmax =
                  s *
                  (d11 * cpu_R[(static_cast<int>((k + 3) + ii) + 169 * k) - 1] -
                   cpu_R[(static_cast<int>((k + 3) + ii) + 169 * (k + 1)) - 1]);
              wkp1 =
                  s *
                  (smax *
                       cpu_R[(static_cast<int>((k + 3) + ii) + 169 * (k + 1)) -
                             1] -
                   cpu_R[(static_cast<int>((k + 3) + ii) + 169 * k) - 1]);
              c = computeEndIdx(
                  static_cast<long>(static_cast<int>((k + 3) + ii)), 169L, 1L);
              for (long i{0L}; i <= c; i++) {
                cpu_R[(static_cast<int>(static_cast<int>((k + 3) + ii) + i) +
                       169 * (static_cast<int>((k + 3) + ii) - 1)) -
                      1] =
                    (cpu_R[(static_cast<int>(static_cast<int>((k + 3) + ii) +
                                             i) +
                            169 * (static_cast<int>((k + 3) + ii) - 1)) -
                           1] -
                     cpu_R[(static_cast<int>(static_cast<int>((k + 3) + ii) +
                                             i) +
                            169 * k) -
                           1] /
                         temp * colmax) -
                    cpu_R[(static_cast<int>(static_cast<int>((k + 3) + ii) +
                                            i) +
                           169 * (k + 1)) -
                          1] /
                        temp * wkp1;
              }
              cpu_R[(static_cast<int>((k + 3) + ii) + 169 * k) - 1] =
                  colmax / temp;
              cpu_R[(static_cast<int>((k + 3) + ii) + 169 * (k + 1)) - 1] =
                  wkp1 / temp;
              R_outdatedOnGpu = true;
            }
          }
          ipiv[k] = -b_p - 1;
          ipiv[k + 1] = -imax - 1;
        }
      }
    }
    native_multi_target_detection_ldl_kernel25<<<dim3(100U, 1U, 1U),
                                                 dim3(288U, 1U, 1U)>>>(*gpu_L);
    L_outdatedOnGpu = false;
    L_outdatedOnCpu = true;
    k = 1;
    while (k <= 169) {
      if (L_outdatedOnGpu) {
        checkCudaError(
            hipMemcpy(*gpu_L, cpu_L, 114244UL, hipMemcpyHostToDevice),
            __FILE__, __LINE__);
      }
      native_multi_target_detection_ldl_kernel26<<<dim3(1U, 1U, 1U),
                                                   dim3(32U, 1U, 1U)>>>(k,
                                                                        *gpu_L);
      L_outdatedOnGpu = false;
      L_outdatedOnCpu = true;
      if (ipiv[k - 1] > 0) {
        b = computeEndIdx(static_cast<long>(k + 1), 169L, 1L);
        for (long ii{0L}; ii <= b; ii++) {
          if (L_outdatedOnCpu) {
            checkCudaError(
                hipMemcpy(cpu_L, *gpu_L, 114244UL, hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          if (R_outdatedOnCpu) {
            checkCudaError(
                hipMemcpy(cpu_R, *gpu_R, 114244UL, hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          R_outdatedOnCpu = false;
          cpu_L[(static_cast<int>((k + 1) + ii) + 169 * (k - 1)) - 1] =
              cpu_R[(static_cast<int>((k + 1) + ii) + 169 * (k - 1)) - 1];
          L_outdatedOnCpu = false;
          L_outdatedOnGpu = true;
        }
        k++;
      } else {
        native_multi_target_detection_ldl_kernel27<<<dim3(1U, 1U, 1U),
                                                     dim3(32U, 1U, 1U)>>>(
            k, *gpu_L);
        b = computeEndIdx(static_cast<long>(k), static_cast<long>(k + 1), 1L);
        mwGetLaunchParameters1D(computeNumIters(b), &grid, &block, 2147483647U);
        if (R_outdatedOnGpu) {
          checkCudaError(
              hipMemcpy(*gpu_R, cpu_R, 114244UL, hipMemcpyHostToDevice),
              __FILE__, __LINE__);
        }
        R_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          native_multi_target_detection_ldl_kernel28<<<grid, block>>>(
              *gpu_R, k, b, *gpu_L);
        }
        k += 2;
      }
    }
    if (ipiv[0] > 0) {
      k = 1;
    } else {
      k = 2;
    }
    while (k + 1 < 169) {
      if (ipiv[k] > 0) {
        cpu_iy = ipiv[k] - 1;
        for (int b_k{0}; b_k < k; b_k++) {
          if (L_outdatedOnCpu) {
            checkCudaError(
                hipMemcpy(cpu_L, *gpu_L, 114244UL, hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          temp = cpu_L[k + b_k * 169];
          cpu_L[k + b_k * 169] = cpu_L[cpu_iy + b_k * 169];
          cpu_L[cpu_iy + b_k * 169] = temp;
          L_outdatedOnCpu = false;
          L_outdatedOnGpu = true;
        }
        k++;
      } else {
        cpu_iy = -ipiv[k] - 1;
        for (int b_k{0}; b_k < k; b_k++) {
          if (L_outdatedOnCpu) {
            checkCudaError(
                hipMemcpy(cpu_L, *gpu_L, 114244UL, hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          temp = cpu_L[k + b_k * 169];
          cpu_L[k + b_k * 169] = cpu_L[cpu_iy + b_k * 169];
          cpu_L[cpu_iy + b_k * 169] = temp;
          L_outdatedOnCpu = false;
          L_outdatedOnGpu = true;
        }
        cpu_iy = -ipiv[k + 1] - 1;
        for (int b_k{0}; b_k < k; b_k++) {
          if (L_outdatedOnCpu) {
            checkCudaError(
                hipMemcpy(cpu_L, *gpu_L, 114244UL, hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          temp = cpu_L[(k + b_k * 169) + 1];
          cpu_L[(k + b_k * 169) + 1] = cpu_L[cpu_iy + b_k * 169];
          cpu_L[cpu_iy + b_k * 169] = temp;
          L_outdatedOnCpu = false;
          L_outdatedOnGpu = true;
        }
        k += 2;
      }
    }
    native_multi_target_detection_ldl_kernel29<<<dim3(100U, 1U, 1U),
                                                 dim3(288U, 1U, 1U)>>>(*gpu_D);
    R_outdatedOnGpu = false;
    p = true;
    k = 0;
    while (k + 1 <= 169) {
      if (ipiv[k] > 0) {
        if (p) {
          checkCudaError(
              hipMemcpy(cpu_D, *gpu_D, 114244UL, hipMemcpyDeviceToHost),
              __FILE__, __LINE__);
        }
        if (R_outdatedOnCpu) {
          checkCudaError(
              hipMemcpy(cpu_R, *gpu_R, 114244UL, hipMemcpyDeviceToHost),
              __FILE__, __LINE__);
        }
        R_outdatedOnCpu = false;
        cpu_D[k + 169 * k] = cpu_R[k + 169 * k];
        p = false;
        R_outdatedOnGpu = true;
        k++;
      } else {
        if (p) {
          checkCudaError(
              hipMemcpy(cpu_D, *gpu_D, 114244UL, hipMemcpyDeviceToHost),
              __FILE__, __LINE__);
        }
        if (R_outdatedOnCpu) {
          checkCudaError(
              hipMemcpy(cpu_R, *gpu_R, 114244UL, hipMemcpyDeviceToHost),
              __FILE__, __LINE__);
        }
        cpu_D[k + 169 * k] = cpu_R[k + 169 * k];
        cpu_D[(k + 169 * (k + 1)) + 1] = cpu_R[(k + 169 * (k + 1)) + 1];
        R_outdatedOnCpu = false;
        cpu_D[(k + 169 * k) + 1] = cpu_R[(k + 169 * k) + 1];
        cpu_D[k + 169 * (k + 1)] = cpu_D[(k + 169 * k) + 1];
        p = false;
        R_outdatedOnGpu = true;
        k += 2;
      }
    }
    k = 168;
    while (k + 1 >= 1) {
      if (ipiv[k] > 0) {
        cpu_iy = ipiv[k] - 1;
        for (int b_k{0}; b_k < 169; b_k++) {
          if (L_outdatedOnCpu) {
            checkCudaError(
                hipMemcpy(cpu_L, *gpu_L, 114244UL, hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          temp = cpu_L[k + b_k * 169];
          cpu_L[k + b_k * 169] = cpu_L[cpu_iy + b_k * 169];
          cpu_L[cpu_iy + b_k * 169] = temp;
          L_outdatedOnCpu = false;
          L_outdatedOnGpu = true;
        }
        k--;
      } else {
        cpu_iy = -ipiv[k] - 1;
        for (int b_k{0}; b_k < 169; b_k++) {
          if (L_outdatedOnCpu) {
            checkCudaError(
                hipMemcpy(cpu_L, *gpu_L, 114244UL, hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          temp = cpu_L[k + b_k * 169];
          cpu_L[k + b_k * 169] = cpu_L[cpu_iy + b_k * 169];
          cpu_L[cpu_iy + b_k * 169] = temp;
          L_outdatedOnCpu = false;
          L_outdatedOnGpu = true;
        }
        cpu_iy = -ipiv[k - 1] - 1;
        for (int b_k{0}; b_k < 169; b_k++) {
          if (L_outdatedOnCpu) {
            checkCudaError(
                hipMemcpy(cpu_L, *gpu_L, 114244UL, hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          temp = cpu_L[(k + b_k * 169) - 1];
          cpu_L[(k + b_k * 169) - 1] = cpu_L[cpu_iy + b_k * 169];
          cpu_L[cpu_iy + b_k * 169] = temp;
          L_outdatedOnCpu = false;
          L_outdatedOnGpu = true;
        }
        k -= 2;
      }
    }
  }
  checkCudaError(hipMemcpy(*gpu_T, cpu_T, 10140UL, hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
  native_multi_target_detection_ldl_kernel4<<<dim3(20U, 1U, 1U),
                                              dim3(128U, 1U, 1U)>>>(*gpu_T,
                                                                    *gpu_W);
  if (L_outdatedOnGpu) {
    checkCudaError(hipMemcpy(*gpu_L, cpu_L, 114244UL, hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  native_multi_target_detection_ldl_kernel5<<<dim3(100U, 1U, 1U),
                                              dim3(288U, 1U, 1U)>>>(*gpu_L,
                                                                    *gpu_R);
  cusolverCheck(hipsolverDnSgetrf_bufferSize(getCuSolverGlobalHandle(), 169, 169,
                                            (float *)&(*gpu_R)[0], 169,
                                            getCuSolverWorkspaceReq()),
                __FILE__, __LINE__);
  setCuSolverWorkspaceTypeSize(4);
  cusolverInitWorkspace();
  cusolverCheck(hipsolverDnSgetrf(
                    getCuSolverGlobalHandle(), 169, 169, (float *)&(*gpu_R)[0],
                    169, static_cast<float *>(getCuSolverWorkspaceBuff()),
                    &(*gpu_IPIV)[0], gpu_info),
                __FILE__, __LINE__);
  checkCudaError(hipMemcpy(&cpu_info, gpu_info, 4UL, hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
  if (cpu_info < 0) {
    native_multi_target_detection_ldl_kernel6<<<dim3(20U, 1U, 1U),
                                                dim3(128U, 1U, 1U)>>>(*gpu_W);
  } else {
    cusolverCheck(hipsolverDnSgetrs(getCuSolverGlobalHandle(), HIPBLAS_OP_N, 169,
                                   15, (float *)&(*gpu_R)[0], 169,
                                   &(*gpu_IPIV)[0], (float *)&(*gpu_W)[0], 169,
                                   gpu_iy),
                  __FILE__, __LINE__);
  }
  if (R_outdatedOnGpu) {
    checkCudaError(hipMemcpy(*gpu_D, cpu_D, 114244UL, hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  cusolverCheck(hipsolverDnSgetrf_bufferSize(getCuSolverGlobalHandle(), 169, 169,
                                            (float *)&(*gpu_D)[0], 169,
                                            getCuSolverWorkspaceReq()),
                __FILE__, __LINE__);
  setCuSolverWorkspaceTypeSize(4);
  cusolverInitWorkspace();
  cusolverCheck(hipsolverDnSgetrf(
                    getCuSolverGlobalHandle(), 169, 169, (float *)&(*gpu_D)[0],
                    169, static_cast<float *>(getCuSolverWorkspaceBuff()),
                    &(*gpu_IPIV)[0], b_gpu_info),
                __FILE__, __LINE__);
  checkCudaError(
      hipMemcpy(&b_cpu_info, b_gpu_info, 4UL, hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  if (b_cpu_info < 0) {
    native_multi_target_detection_ldl_kernel7<<<dim3(20U, 1U, 1U),
                                                dim3(128U, 1U, 1U)>>>(*gpu_W);
  } else {
    cusolverCheck(hipsolverDnSgetrs(getCuSolverGlobalHandle(), HIPBLAS_OP_N, 169,
                                   15, (float *)&(*gpu_D)[0], 169,
                                   &(*gpu_IPIV)[0], (float *)&(*gpu_W)[0], 169,
                                   gpu_iy),
                  __FILE__, __LINE__);
  }
  native_multi_target_detection_ldl_kernel8<<<dim3(100U, 1U, 1U),
                                              dim3(288U, 1U, 1U)>>>(*gpu_L,
                                                                    *gpu_R);
  cusolverCheck(hipsolverDnSgetrf_bufferSize(getCuSolverGlobalHandle(), 169, 169,
                                            (float *)&(*gpu_R)[0], 169,
                                            getCuSolverWorkspaceReq()),
                __FILE__, __LINE__);
  setCuSolverWorkspaceTypeSize(4);
  cusolverInitWorkspace();
  cusolverCheck(hipsolverDnSgetrf(
                    getCuSolverGlobalHandle(), 169, 169, (float *)&(*gpu_R)[0],
                    169, static_cast<float *>(getCuSolverWorkspaceBuff()),
                    &(*gpu_IPIV)[0], c_gpu_info),
                __FILE__, __LINE__);
  checkCudaError(
      hipMemcpy(&c_cpu_info, c_gpu_info, 4UL, hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  if (c_cpu_info < 0) {
    native_multi_target_detection_ldl_kernel9<<<dim3(20U, 1U, 1U),
                                                dim3(128U, 1U, 1U)>>>(*gpu_W);
  } else {
    cusolverCheck(hipsolverDnSgetrs(getCuSolverGlobalHandle(), HIPBLAS_OP_N, 169,
                                   15, (float *)&(*gpu_R)[0], 169,
                                   &(*gpu_IPIV)[0], (float *)&(*gpu_W)[0], 169,
                                   gpu_iy),
                  __FILE__, __LINE__);
  }
  //  TIMING_3

  end = clock();
  hipDeviceSynchronize();
  printf("LDL 1: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();


  native_multi_target_detection_ldl_kernel10<<<dim3(20U, 1U, 1U),
                                               dim3(128U, 1U, 1U)>>>(*gpu_T,
                                                                     *b_gpu_a);
  temp = 1.0F;
  smax = 0.0F;
  cublasCheck(hipblasSgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, 15,
                          15, 169, (float *)&temp, (float *)&(*b_gpu_a)[0], 15,
                          (float *)&(*gpu_W)[0], 169, (float *)&smax,
                          (float *)&(*gpu_t2)[0], 15),
              __FILE__, __LINE__);
  R_outdatedOnGpu = false;
  R_outdatedOnCpu = true;
  //  TIMING_4

  end = clock();
  hipDeviceSynchronize();
  printf("MUL 1: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();

  //  Alternative: t3 = t2 \ C;
  p = true;
  for (k = 0; k < 225; k++) {
    if (p) {
      if (R_outdatedOnCpu) {
        checkCudaError(
            hipMemcpy(cpu_t2, *gpu_t2, 900UL, hipMemcpyDeviceToHost),
            __FILE__, __LINE__);
      }
      R_outdatedOnCpu = false;
      temp = cpu_t2[k];
      if (std::isinf(temp) || std::isnan(temp)) {
        p = false;
      }
    } else {
      p = false;
    }
  }
  if (!p) {
    native_multi_target_detection_ldl_kernel11<<<dim3(2U, 1U, 1U),
                                                 dim3(128U, 1U, 1U)>>>(
        *b_gpu_D, *b_gpu_L);
    L_outdatedOnGpu = false;
    R_outdatedOnGpu = false;
  } else {
    int b_ipiv[15];
    for (k = 0; k + 1 <= 15; k += kstep) {
      kstep = 1;
      b_p = k;
      if (R_outdatedOnCpu) {
        checkCudaError(
            hipMemcpy(cpu_t2, *gpu_t2, 900UL, hipMemcpyDeviceToHost),
            __FILE__, __LINE__);
      }
      R_outdatedOnCpu = false;
      temp = std::abs(cpu_t2[k + 15 * k]);
      if (k + 1 < 15) {
        cpu_iy = (k * 15 + k) + 1;
        n = 12 - k;
        ix = 1;
        if (14 - k > 1) {
          smax = std::abs(cpu_t2[cpu_iy]);
          for (int b_k{0}; b_k <= n; b_k++) {
            s = std::abs(cpu_t2[(cpu_iy + b_k) + 1]);
            if (s > smax) {
              ix = b_k + 2;
              smax = s;
            }
          }
        }
        imax = k + ix;
        colmax = std::abs(cpu_t2[imax + 15 * k]);
      } else {
        imax = 14;
        colmax = 0.0F;
      }
      if (std::fmax(temp, colmax) == 0.0F) {
        imax = k;
        b_ipiv[k] = imax + 1;
      } else {
        if (!(temp < 0.640388191F * colmax)) {
          imax = k;
        } else {
          do {
            exitg1 = 0;
            if (imax + 1 != k + 1) {
              cpu_iy = k * 15 + imax;
              n = imax - k;
              if (n < 1) {
                ix = 0;
              } else {
                ix = 1;
                if (n > 1) {
                  smax = std::abs(cpu_t2[cpu_iy]);
                  for (int b_k{0}; b_k <= n - 2; b_k++) {
                    s = std::abs(cpu_t2[cpu_iy + (b_k + 1) * 15]);
                    if (s > smax) {
                      ix = b_k + 2;
                      smax = s;
                    }
                  }
                }
              }
              jmax = k + ix;
              temp = std::abs(cpu_t2[imax + 15 * (jmax - 1)]);
            } else {
              temp = 0.0F;
              jmax = k;
            }
            if (imax + 1 < 15) {
              cpu_iy = (imax * 15 + imax) + 1;
              n = 12 - imax;
              ix = 2;
              if (14 - imax > 1) {
                smax = std::abs(cpu_t2[cpu_iy]);
                for (int b_k{0}; b_k <= n; b_k++) {
                  s = std::abs(cpu_t2[(cpu_iy + b_k) + 1]);
                  if (s > smax) {
                    ix = b_k + 3;
                    smax = s;
                  }
                }
              }
              cpu_iy = imax + ix;
              smax = std::abs(cpu_t2[(cpu_iy + 15 * imax) - 1]);
              if (smax > temp) {
                temp = smax;
                jmax = cpu_iy;
              }
            }
            if (!(std::abs(cpu_t2[imax + 15 * imax]) < 0.640388191F * temp)) {
              exitg1 = 1;
            } else if ((b_p + 1 == jmax) || (temp <= colmax)) {
              kstep = 2;
              exitg1 = 1;
            } else {
              b_p = imax;
              colmax = temp;
              imax = jmax - 1;
            }
          } while (exitg1 == 0);
        }
        jmax = (k + kstep) - 1;
        if ((kstep == 2) && (b_p + 1 != k + 1)) {
          if (b_p + 1 < 15) {
            ix = (k * 15 + b_p) + 1;
            cpu_iy = (b_p * 15 + b_p) + 1;
            n = 13 - b_p;
            for (int b_k{0}; b_k <= n; b_k++) {
              temp = cpu_t2[ix + b_k];
              cpu_t2[ix + b_k] = cpu_t2[cpu_iy + b_k];
              cpu_t2[cpu_iy + b_k] = temp;
            }
          }
          b = computeEndIdx(static_cast<long>(k + 2), static_cast<long>(b_p),
                            1L);
          for (long ii{0L}; ii <= b; ii++) {
            temp = cpu_t2[(static_cast<int>((k + 2) + ii) + 15 * k) - 1];
            cpu_t2[(static_cast<int>((k + 2) + ii) + 15 * k) - 1] =
                cpu_t2[b_p + 15 * (static_cast<int>((k + 2) + ii) - 1)];
            cpu_t2[b_p + 15 * (static_cast<int>((k + 2) + ii) - 1)] = temp;
          }
          temp = cpu_t2[k + 15 * k];
          cpu_t2[k + 15 * k] = cpu_t2[b_p + 15 * b_p];
          cpu_t2[b_p + 15 * b_p] = temp;
          R_outdatedOnGpu = true;
        }
        if (imax + 1 != jmax + 1) {
          if (imax + 1 < 15) {
            ix = (jmax * 15 + imax) + 1;
            cpu_iy = (imax * 15 + imax) + 1;
            n = 13 - imax;
            for (int b_k{0}; b_k <= n; b_k++) {
              temp = cpu_t2[ix + b_k];
              cpu_t2[ix + b_k] = cpu_t2[cpu_iy + b_k];
              cpu_t2[cpu_iy + b_k] = temp;
            }
          }
          b = computeEndIdx(static_cast<long>(jmax + 2),
                            static_cast<long>(imax), 1L);
          for (long ii{0L}; ii <= b; ii++) {
            temp = cpu_t2[(static_cast<int>((jmax + 2) + ii) + 15 * jmax) - 1];
            cpu_t2[(static_cast<int>((jmax + 2) + ii) + 15 * jmax) - 1] =
                cpu_t2[imax + 15 * (static_cast<int>((jmax + 2) + ii) - 1)];
            cpu_t2[imax + 15 * (static_cast<int>((jmax + 2) + ii) - 1)] = temp;
          }
          temp = cpu_t2[jmax + 15 * jmax];
          cpu_t2[jmax + 15 * jmax] = cpu_t2[imax + 15 * imax];
          cpu_t2[imax + 15 * imax] = temp;
          R_outdatedOnGpu = true;
          if (kstep == 2) {
            temp = cpu_t2[(k + 15 * k) + 1];
            cpu_t2[(k + 15 * k) + 1] = cpu_t2[imax + 15 * k];
            cpu_t2[imax + 15 * k] = temp;
          }
        }
        if (kstep == 1) {
          if (k + 1 < 15) {
            if (std::abs(cpu_t2[k + 15 * k]) >= 9.09494702E-13F) {
              d11 = 1.0F / cpu_t2[k + 15 * k];
              b = computeEndIdx(static_cast<long>(k + 2), 15L, 1L);
              for (long ii{0L}; ii <= b; ii++) {
                if (cpu_t2[(static_cast<int>((k + 2) + ii) + 15 * k) - 1] !=
                    0.0F) {
                  temp = -d11 *
                         cpu_t2[(static_cast<int>((k + 2) + ii) + 15 * k) - 1];
                  cpu_t2[(static_cast<int>((k + 2) + ii) +
                          15 * (static_cast<int>((k + 2) + ii) - 1)) -
                         1] +=
                      temp *
                      cpu_t2[(static_cast<int>((k + 2) + ii) + 15 * k) - 1];
                  R_outdatedOnGpu = true;
                  ix = static_cast<int>((k + 2) + ii) + 1;
                  c = computeEndIdx(static_cast<long>(ix), 15L, 1L);
                  for (long i{0L}; i <= c; i++) {
                    cpu_t2[(static_cast<int>(ix + i) +
                            15 * (static_cast<int>((k + 2) + ii) - 1)) -
                           1] +=
                        cpu_t2[(static_cast<int>(ix + i) + 15 * k) - 1] * temp;
                  }
                }
              }
              cpu_iy = k * 15 + k;
              b = computeEndIdx(static_cast<long>(cpu_iy + 2),
                                static_cast<long>((cpu_iy - k) + 15), 1L);
              mwGetLaunchParameters1D(computeNumIters(b), &grid, &block,
                                      2147483647U);
              if (R_outdatedOnGpu) {
                checkCudaError(
                    hipMemcpy(*gpu_t2, cpu_t2, 900UL, hipMemcpyHostToDevice),
                    __FILE__, __LINE__);
              }
              validLaunchParams = mwValidateLaunchParameters(grid, block);
              if (validLaunchParams) {
                native_multi_target_detection_ldl_kernel18<<<grid, block>>>(
                    d11, cpu_iy + 2, b, *gpu_t2);
              }
              R_outdatedOnGpu = false;
              R_outdatedOnCpu = true;
            } else {
              d11 = cpu_t2[k + 15 * k];
              b = computeEndIdx(static_cast<long>(k + 2), 15L, 1L);
              for (long ii{0L}; ii <= b; ii++) {
                cpu_t2[(static_cast<int>((k + 2) + ii) + 15 * k) - 1] /= d11;
                R_outdatedOnGpu = true;
              }
              b = computeEndIdx(static_cast<long>(k + 2), 15L, 1L);
              for (long ii{0L}; ii <= b; ii++) {
                if (cpu_t2[(static_cast<int>((k + 2) + ii) + 15 * k) - 1] !=
                    0.0F) {
                  temp = -d11 *
                         cpu_t2[(static_cast<int>((k + 2) + ii) + 15 * k) - 1];
                  cpu_t2[(static_cast<int>((k + 2) + ii) +
                          15 * (static_cast<int>((k + 2) + ii) - 1)) -
                         1] +=
                      temp *
                      cpu_t2[(static_cast<int>((k + 2) + ii) + 15 * k) - 1];
                  R_outdatedOnGpu = true;
                  ix = static_cast<int>((k + 2) + ii) + 1;
                  c = computeEndIdx(static_cast<long>(ix), 15L, 1L);
                  for (long i{0L}; i <= c; i++) {
                    cpu_t2[(static_cast<int>(ix + i) +
                            15 * (static_cast<int>((k + 2) + ii) - 1)) -
                           1] +=
                        cpu_t2[(static_cast<int>(ix + i) + 15 * k) - 1] * temp;
                  }
                }
              }
            }
          }
          b_ipiv[k] = imax + 1;
        } else {
          if (k + 1 < 14) {
            temp = cpu_t2[(k + 15 * k) + 1];
            d11 = cpu_t2[(k + 15 * (k + 1)) + 1] / cpu_t2[(k + 15 * k) + 1];
            smax = cpu_t2[k + 15 * k] / cpu_t2[(k + 15 * k) + 1];
            s = 1.0F / (d11 * smax - 1.0F);
            b = computeEndIdx(static_cast<long>(k + 3), 15L, 1L);
            for (long ii{0L}; ii <= b; ii++) {
              colmax =
                  s *
                  (d11 * cpu_t2[(static_cast<int>((k + 3) + ii) + 15 * k) - 1] -
                   cpu_t2[(static_cast<int>((k + 3) + ii) + 15 * (k + 1)) - 1]);
              wkp1 =
                  s *
                  (smax *
                       cpu_t2[(static_cast<int>((k + 3) + ii) + 15 * (k + 1)) -
                              1] -
                   cpu_t2[(static_cast<int>((k + 3) + ii) + 15 * k) - 1]);
              c = computeEndIdx(
                  static_cast<long>(static_cast<int>((k + 3) + ii)), 15L, 1L);
              for (long i{0L}; i <= c; i++) {
                cpu_t2[(static_cast<int>(static_cast<int>((k + 3) + ii) + i) +
                        15 * (static_cast<int>((k + 3) + ii) - 1)) -
                       1] =
                    (cpu_t2[(static_cast<int>(static_cast<int>((k + 3) + ii) +
                                              i) +
                             15 * (static_cast<int>((k + 3) + ii) - 1)) -
                            1] -
                     cpu_t2[(static_cast<int>(static_cast<int>((k + 3) + ii) +
                                              i) +
                             15 * k) -
                            1] /
                         temp * colmax) -
                    cpu_t2[(static_cast<int>(static_cast<int>((k + 3) + ii) +
                                             i) +
                            15 * (k + 1)) -
                           1] /
                        temp * wkp1;
              }
              cpu_t2[(static_cast<int>((k + 3) + ii) + 15 * k) - 1] =
                  colmax / temp;
              cpu_t2[(static_cast<int>((k + 3) + ii) + 15 * (k + 1)) - 1] =
                  wkp1 / temp;
              R_outdatedOnGpu = true;
            }
          }
          b_ipiv[k] = -b_p - 1;
          b_ipiv[k + 1] = -imax - 1;
        }
      }
    }
    native_multi_target_detection_ldl_kernel19<<<dim3(2U, 1U, 1U),
                                                 dim3(128U, 1U, 1U)>>>(
        *b_gpu_L);
    L_outdatedOnGpu = false;
    L_outdatedOnCpu = true;
    k = 1;
    while (k <= 15) {
      if (L_outdatedOnGpu) {
        checkCudaError(
            hipMemcpy(*b_gpu_L, b_cpu_L, 900UL, hipMemcpyHostToDevice),
            __FILE__, __LINE__);
      }
      native_multi_target_detection_ldl_kernel20<<<dim3(1U, 1U, 1U),
                                                   dim3(32U, 1U, 1U)>>>(
          k, *b_gpu_L);
      L_outdatedOnGpu = false;
      L_outdatedOnCpu = true;
      if (b_ipiv[k - 1] > 0) {
        b = computeEndIdx(static_cast<long>(k + 1), 15L, 1L);
        for (long ii{0L}; ii <= b; ii++) {
          if (L_outdatedOnCpu) {
            checkCudaError(
                hipMemcpy(b_cpu_L, *b_gpu_L, 900UL, hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          if (R_outdatedOnCpu) {
            checkCudaError(
                hipMemcpy(cpu_t2, *gpu_t2, 900UL, hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          R_outdatedOnCpu = false;
          b_cpu_L[(static_cast<int>((k + 1) + ii) + 15 * (k - 1)) - 1] =
              cpu_t2[(static_cast<int>((k + 1) + ii) + 15 * (k - 1)) - 1];
          L_outdatedOnCpu = false;
          L_outdatedOnGpu = true;
        }
        k++;
      } else {
        native_multi_target_detection_ldl_kernel21<<<dim3(1U, 1U, 1U),
                                                     dim3(32U, 1U, 1U)>>>(
            k, *b_gpu_L);
        b = computeEndIdx(static_cast<long>(k), static_cast<long>(k + 1), 1L);
        mwGetLaunchParameters1D(computeNumIters(b), &grid, &block, 2147483647U);
        if (R_outdatedOnGpu) {
          checkCudaError(
              hipMemcpy(*gpu_t2, cpu_t2, 900UL, hipMemcpyHostToDevice),
              __FILE__, __LINE__);
        }
        R_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          native_multi_target_detection_ldl_kernel22<<<grid, block>>>(
              *gpu_t2, k, b, *b_gpu_L);
        }
        k += 2;
      }
    }
    if (b_ipiv[0] > 0) {
      k = 1;
    } else {
      k = 2;
    }
    while (k + 1 < 15) {
      if (b_ipiv[k] > 0) {
        cpu_iy = b_ipiv[k] - 1;
        for (int b_k{0}; b_k < k; b_k++) {
          if (L_outdatedOnCpu) {
            checkCudaError(
                hipMemcpy(b_cpu_L, *b_gpu_L, 900UL, hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          temp = b_cpu_L[k + b_k * 15];
          b_cpu_L[k + b_k * 15] = b_cpu_L[cpu_iy + b_k * 15];
          b_cpu_L[cpu_iy + b_k * 15] = temp;
          L_outdatedOnCpu = false;
          L_outdatedOnGpu = true;
        }
        k++;
      } else {
        cpu_iy = -b_ipiv[k] - 1;
        for (int b_k{0}; b_k < k; b_k++) {
          if (L_outdatedOnCpu) {
            checkCudaError(
                hipMemcpy(b_cpu_L, *b_gpu_L, 900UL, hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          temp = b_cpu_L[k + b_k * 15];
          b_cpu_L[k + b_k * 15] = b_cpu_L[cpu_iy + b_k * 15];
          b_cpu_L[cpu_iy + b_k * 15] = temp;
          L_outdatedOnCpu = false;
          L_outdatedOnGpu = true;
        }
        cpu_iy = -b_ipiv[k + 1] - 1;
        for (int b_k{0}; b_k < k; b_k++) {
          if (L_outdatedOnCpu) {
            checkCudaError(
                hipMemcpy(b_cpu_L, *b_gpu_L, 900UL, hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          temp = b_cpu_L[(k + b_k * 15) + 1];
          b_cpu_L[(k + b_k * 15) + 1] = b_cpu_L[cpu_iy + b_k * 15];
          b_cpu_L[cpu_iy + b_k * 15] = temp;
          L_outdatedOnCpu = false;
          L_outdatedOnGpu = true;
        }
        k += 2;
      }
    }
    native_multi_target_detection_ldl_kernel23<<<dim3(2U, 1U, 1U),
                                                 dim3(128U, 1U, 1U)>>>(
        *b_gpu_D);
    R_outdatedOnGpu = false;
    p = true;
    k = 0;
    while (k + 1 <= 15) {
      if (b_ipiv[k] > 0) {
        if (p) {
          checkCudaError(
              hipMemcpy(b_cpu_D, *b_gpu_D, 900UL, hipMemcpyDeviceToHost),
              __FILE__, __LINE__);
        }
        if (R_outdatedOnCpu) {
          checkCudaError(
              hipMemcpy(cpu_t2, *gpu_t2, 900UL, hipMemcpyDeviceToHost),
              __FILE__, __LINE__);
        }
        R_outdatedOnCpu = false;
        b_cpu_D[k + 15 * k] = cpu_t2[k + 15 * k];
        p = false;
        R_outdatedOnGpu = true;
        k++;
      } else {
        if (p) {
          checkCudaError(
              hipMemcpy(b_cpu_D, *b_gpu_D, 900UL, hipMemcpyDeviceToHost),
              __FILE__, __LINE__);
        }
        if (R_outdatedOnCpu) {
          checkCudaError(
              hipMemcpy(cpu_t2, *gpu_t2, 900UL, hipMemcpyDeviceToHost),
              __FILE__, __LINE__);
        }
        b_cpu_D[k + 15 * k] = cpu_t2[k + 15 * k];
        b_cpu_D[(k + 15 * (k + 1)) + 1] = cpu_t2[(k + 15 * (k + 1)) + 1];
        R_outdatedOnCpu = false;
        b_cpu_D[(k + 15 * k) + 1] = cpu_t2[(k + 15 * k) + 1];
        b_cpu_D[k + 15 * (k + 1)] = b_cpu_D[(k + 15 * k) + 1];
        p = false;
        R_outdatedOnGpu = true;
        k += 2;
      }
    }
    k = 14;
    while (k + 1 >= 1) {
      if (b_ipiv[k] > 0) {
        cpu_iy = b_ipiv[k] - 1;
        for (int b_k{0}; b_k < 15; b_k++) {
          if (L_outdatedOnCpu) {
            checkCudaError(
                hipMemcpy(b_cpu_L, *b_gpu_L, 900UL, hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          temp = b_cpu_L[k + b_k * 15];
          b_cpu_L[k + b_k * 15] = b_cpu_L[cpu_iy + b_k * 15];
          b_cpu_L[cpu_iy + b_k * 15] = temp;
          L_outdatedOnCpu = false;
          L_outdatedOnGpu = true;
        }
        k--;
      } else {
        cpu_iy = -b_ipiv[k] - 1;
        for (int b_k{0}; b_k < 15; b_k++) {
          if (L_outdatedOnCpu) {
            checkCudaError(
                hipMemcpy(b_cpu_L, *b_gpu_L, 900UL, hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          temp = b_cpu_L[k + b_k * 15];
          b_cpu_L[k + b_k * 15] = b_cpu_L[cpu_iy + b_k * 15];
          b_cpu_L[cpu_iy + b_k * 15] = temp;
          L_outdatedOnCpu = false;
          L_outdatedOnGpu = true;
        }
        cpu_iy = -b_ipiv[k - 1] - 1;
        for (int b_k{0}; b_k < 15; b_k++) {
          if (L_outdatedOnCpu) {
            checkCudaError(
                hipMemcpy(b_cpu_L, *b_gpu_L, 900UL, hipMemcpyDeviceToHost),
                __FILE__, __LINE__);
          }
          temp = b_cpu_L[(k + b_k * 15) - 1];
          b_cpu_L[(k + b_k * 15) - 1] = b_cpu_L[cpu_iy + b_k * 15];
          b_cpu_L[cpu_iy + b_k * 15] = temp;
          L_outdatedOnCpu = false;
          L_outdatedOnGpu = true;
        }
        k -= 2;
      }
    }
  }
  checkCudaError(hipMemcpy(*gpu_C, cpu_C, 180UL, hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
  native_multi_target_detection_ldl_kernel12<<<dim3(1U, 1U, 1U),
                                               dim3(64U, 1U, 1U)>>>(*gpu_C,
                                                                    *b_gpu_W);
  if (L_outdatedOnGpu) {
    checkCudaError(hipMemcpy(*b_gpu_L, b_cpu_L, 900UL, hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  native_multi_target_detection_ldl_kernel13<<<dim3(2U, 1U, 1U),
                                               dim3(128U, 1U, 1U)>>>(*b_gpu_L,
                                                                     *gpu_t2);
  cusolverCheck(hipsolverDnSgetrf_bufferSize(getCuSolverGlobalHandle(), 15, 15,
                                            (float *)&(*gpu_t2)[0], 15,
                                            getCuSolverWorkspaceReq()),
                __FILE__, __LINE__);
  setCuSolverWorkspaceTypeSize(4);
  cusolverInitWorkspace();
  cusolverCheck(hipsolverDnSgetrf(
                    getCuSolverGlobalHandle(), 15, 15, (float *)&(*gpu_t2)[0],
                    15, static_cast<float *>(getCuSolverWorkspaceBuff()),
                    &(*b_gpu_IPIV)[0], d_gpu_info),
                __FILE__, __LINE__);
  checkCudaError(
      hipMemcpy(&d_cpu_info, d_gpu_info, 4UL, hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  if (d_cpu_info < 0) {
    native_multi_target_detection_ldl_kernel14<<<dim3(1U, 1U, 1U),
                                                 dim3(64U, 1U, 1U)>>>(*b_gpu_W);
  } else {
    cusolverCheck(hipsolverDnSgetrs(getCuSolverGlobalHandle(), HIPBLAS_OP_N, 15,
                                   3, (float *)&(*gpu_t2)[0], 15,
                                   &(*b_gpu_IPIV)[0], (float *)&(*b_gpu_W)[0],
                                   15, gpu_iy),
                  __FILE__, __LINE__);
  }
  if (R_outdatedOnGpu) {
    checkCudaError(hipMemcpy(*b_gpu_D, b_cpu_D, 900UL, hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
  }
  cusolverCheck(hipsolverDnSgetrf_bufferSize(getCuSolverGlobalHandle(), 15, 15,
                                            (float *)&(*b_gpu_D)[0], 15,
                                            getCuSolverWorkspaceReq()),
                __FILE__, __LINE__);
  setCuSolverWorkspaceTypeSize(4);
  cusolverInitWorkspace();
  cusolverCheck(hipsolverDnSgetrf(
                    getCuSolverGlobalHandle(), 15, 15, (float *)&(*b_gpu_D)[0],
                    15, static_cast<float *>(getCuSolverWorkspaceBuff()),
                    &(*b_gpu_IPIV)[0], e_gpu_info),
                __FILE__, __LINE__);
  checkCudaError(
      hipMemcpy(&e_cpu_info, e_gpu_info, 4UL, hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  if (e_cpu_info < 0) {
    native_multi_target_detection_ldl_kernel15<<<dim3(1U, 1U, 1U),
                                                 dim3(64U, 1U, 1U)>>>(*b_gpu_W);
  } else {
    cusolverCheck(hipsolverDnSgetrs(getCuSolverGlobalHandle(), HIPBLAS_OP_N, 15,
                                   3, (float *)&(*b_gpu_D)[0], 15,
                                   &(*b_gpu_IPIV)[0], (float *)&(*b_gpu_W)[0],
                                   15, gpu_iy),
                  __FILE__, __LINE__);
  }
  native_multi_target_detection_ldl_kernel16<<<dim3(2U, 1U, 1U),
                                               dim3(128U, 1U, 1U)>>>(*b_gpu_L,
                                                                     *gpu_t2);
  cusolverCheck(hipsolverDnSgetrf_bufferSize(getCuSolverGlobalHandle(), 15, 15,
                                            (float *)&(*gpu_t2)[0], 15,
                                            getCuSolverWorkspaceReq()),
                __FILE__, __LINE__);
  setCuSolverWorkspaceTypeSize(4);
  cusolverInitWorkspace();
  cusolverCheck(hipsolverDnSgetrf(
                    getCuSolverGlobalHandle(), 15, 15, (float *)&(*gpu_t2)[0],
                    15, static_cast<float *>(getCuSolverWorkspaceBuff()),
                    &(*b_gpu_IPIV)[0], f_gpu_info),
                __FILE__, __LINE__);
  checkCudaError(
      hipMemcpy(&f_cpu_info, f_gpu_info, 4UL, hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  if (f_cpu_info < 0) {
    native_multi_target_detection_ldl_kernel17<<<dim3(1U, 1U, 1U),
                                                 dim3(64U, 1U, 1U)>>>(*b_gpu_W);
  } else {
    cusolverCheck(hipsolverDnSgetrs(getCuSolverGlobalHandle(), HIPBLAS_OP_N, 15,
                                   3, (float *)&(*gpu_t2)[0], 15,
                                   &(*b_gpu_IPIV)[0], (float *)&(*b_gpu_W)[0],
                                   15, gpu_iy),
                  __FILE__, __LINE__);
  }
  //  TIMING_5

  end = clock();
  hipDeviceSynchronize();
  printf("LDL 2: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();


  //  Compute the weighting matrix W
  temp = 1.0F;
  smax = 0.0F;
  cublasCheck(hipblasSgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                          169, 3, 15, (float *)&temp, (float *)&(*gpu_W)[0],
                          169, (float *)&(*b_gpu_W)[0], 15, (float *)&smax,
                          (float *)&(*c_gpu_W)[0], 169),
              __FILE__, __LINE__);
  //  TIMING_6

  end = clock();
  hipDeviceSynchronize();
  printf("MUL 2: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();

  checkCudaError(hipMemcpy(cpu_W, *c_gpu_W, 2028UL, hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);

  end = clock();
  hipDeviceSynchronize();
  printf("Copy back: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();


  checkCudaError(mwCudaFree(*gpu_T), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_C), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_X), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*c_gpu_W), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_a), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_R), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_L), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_D), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_W), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_IPIV), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(gpu_iy), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(gpu_info), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(b_gpu_info), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(c_gpu_info), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*b_gpu_a), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_t2), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*b_gpu_L), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*b_gpu_D), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*b_gpu_W), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*b_gpu_IPIV), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(d_gpu_info), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(e_gpu_info), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(f_gpu_info), __FILE__, __LINE__);

  end = clock();
  hipDeviceSynchronize();
  printf("Free: %lf seconds\n", ((double) (end - start)) / CLOCKS_PER_SEC);
  start = clock();

}

//
// File trailer for native_multi_target_detection_ldl.cu
//
// [EOF]
//
